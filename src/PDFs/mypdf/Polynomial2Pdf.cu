#include "hip/hip_runtime.h"
#include <goofit/PDFs/mypdf/Polynomial2Pdf.h>
#include <goofit/Variable.h>

namespace GooFit {

__device__ fptype device_Polynomial2(fptype *evt, fptype *p, unsigned int *indices) {
    // Structure is nP lowestdegree c1 c2 c3 nO o1

    int numParams    = (indices[0]) + 1;
    int lowestDegree = (indices[1]);

    fptype x   = evt[(indices[2 + (indices[0])])];
    fptype ret = 0;

    for(int i = 2; i < numParams; ++i) {
        ret += (p[(indices[i])]) * pow(x, lowestDegree + i - 2);
    }

    return ret*ret;
}

__device__ fptype device_OffsetPolynomial2(fptype *evt, fptype *p, unsigned int *indices) {
    int numParams    = (indices[0]);
    int lowestDegree = (indices[1]);

    fptype x = evt[(indices[2 + numParams])];
    x -= (p[(indices[numParams])]);
    fptype ret = 0;

    for(int i = 2; i < numParams; ++i) {
        ret += (p[(indices[i])]) * pow(x, lowestDegree + i - 2);
    }

    return ret*ret;
}

__device__ fptype device_MultiPolynomial2(fptype *evt, fptype *p, unsigned int *indices) {
    int numObservables = (indices[(indices[0]) + 1]);
    int maxDegree      = (indices[1]);
    if (numObservables!=3) {
     printf("device_Polynomial2 error: Number of Observables should be = 3!!! numObservables = %d\n",numObservables);
    
     return -100;
    }
 
    fptype x    = (evt[(indices[2 + (indices[0]) ])]); // x, y, z...
    x -= (p[(indices[2])]); ;
    fptype y    = (evt[(indices[2 + (indices[0]) + 1])]); // x, y, z...
    y -= (p[(indices[3])]); ;
    fptype z    = (evt[(indices[2 + (indices[0]) + 2])]); // x, y, z...
    z -= (p[(indices[4])]); ;
    
       int ipar =2 + numObservables;
       int kk = 0;
       int ii = 0;
       int jj = 0;
       fptype func =0;
       for(int i = 0; i <= maxDegree ; ++i) {
       jj = 0;
         for(int j = i; j <= maxDegree ; ++j) {
//	  std::cout<<"func = par["<<ipar<<"]*x^"<<kk<<"*y^"<<jj<<std::endl;
          ii = 0;
          for(int k = j; k <= maxDegree ; ++k) {
//            if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  	    printf("polynomial2  par = %f	x^%d*y^%d*z^%d = %d\n",(p[(indices[ipar])]),ii,jj,kk);
//            } 
//	   std::cout<<"func = par["<<ipar<<"]*x^"<<ii<<"*y^"<<jj<<"*z^"<<kk<<std::endl;
           func +=(p[(indices[ipar])])*pow(x,ii)*pow(y,jj)*pow(z,kk);

//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	printf("polynomial2 MaxDegree=%d coefficient = %f   number = %d\n",maxDegree,(p[(indices[ipar])]),ipar-2-2*numObservables);
//        } 
	   
	   ipar++;
           ii = (jj+kk+ii<maxDegree?++ii:0);
	  }
          jj = (jj+kk+ii<maxDegree?++jj:0);
         }
         kk= (jj+kk+ii<maxDegree?++kk:0);
       }
       return  func*func;
 
 
 }

/* __device__ fptype device_MultiPolynomial2(fptype *evt, fptype *p, unsigned int *indices) {
    // Structure is nP, maxDegree, offset1, offset2, ..., coeff1, coeff2, ..., nO, o1, o2, ...

    int numObservables = (indices[(indices[0]) + 1]);
    int maxDegree      = (indices[1]) + 1;
    // Only appears in construction (maxDegree + 1) or (x > maxDegree), so
    // may as well add the one and use >= instead.

    // Technique is to iterate over the full n-dimensional box, skipping matrix elements
    // whose sum of indices is greater than maxDegree. Notice that this is increasingly
    // inefficient as n grows, since a larger proportion of boxes will be skipped.
    int numBoxes = 1;
    
    for(int i = 0; i < numObservables; ++i)
        numBoxes *= maxDegree;

    int coeffNumber = 2 + numObservables; // Index of first coefficient is 2 + nO, not 1 + nO, due to maxDegree. (nO
                                          // comes from offsets.)
    fptype ret = (p[(indices[coeffNumber++])]); // Coefficient of constant term.
    fptype intg= ret*4*atan2(0.0, -1.0);

    for(int i = 1; i < numBoxes;
        ++i) { // Notice skip of inmost 'box' in the pyramid, corresponding to all powers zero, already accounted for.
        fptype currTerm  = 1;
        int currIndex    = i;
        int sumOfIndices = 0;
	fptype currIntg  = 1;
        fptype x1        = 0;
        fptype x2        = 0;

        // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
        // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
        // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
        // printf("[%i, %i] Start box %i %f %f:\n", BLOCKIDX, THREADIDX, i, ret, evt[8]);
        for(int j = 0; j < numObservables; ++j) {
	    if(j<=1){
	     x1=-1;
	     x2= 1;
	    }
	    if(j>1){
	     x1=0;
	     x2=atan2(0.0, -1.0);
	    }
            fptype x      = (evt[(indices[2 + (indices[0]) + j])]); // x, y, z...
            fptype offset = (p[(indices[2 + j])]);                          // x0, y0, z0...
            x -= offset;
            int currPower = currIndex % maxDegree;
            currIndex /= maxDegree;
            currTerm *= pow(x, currPower);
            sumOfIndices += currPower;
	    currIntg *= (pow(x2, currPower+1)-pow(x1, currPower+1))/(currPower+1); 
            // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
            // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
            // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
            // printf("  [%f -> %f^%i = %f] (%i %i) \n", evt[indices[2 + indices[0] + j]], x, currPower, pow(x,
            // currPower), sumOfIndices, indices[2 + indices[0] + j]);
        }

        // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
        // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
        // printf(") End box %i\n", i);
        // All threads should hit this at the same time and with the same result. No branching.
        if(sumOfIndices >= maxDegree)
            continue;

        fptype coefficient = (p[(indices[coeffNumber++])]); // Coefficient from MINUIT
//         if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//          printf("Polynomial2 MaxDegree=%d coefficient = %f   number = %d\n",maxDegree,coefficient,coeffNumber-3-numObservables);
// 	}
        // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
        // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
        // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
        // printf("Box %i contributes %f * %f = %f -> %f\n", i, currTerm, p[indices[coeffNumber - 1]],
        // coefficient*currTerm, (ret + coefficient*currTerm));
        currTerm *= coefficient;
        currIntg *= coefficient;
        ret += currTerm;
	intg+= currIntg;
    }

    // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
    // printf("Final Polynomial2: %f\n", ret);

// if (0 > ret/(intg)) ret = 0;
// if (0 > ret) ret = -ret;
// PADUL!!!
// if (ret/(intg)>1) printf("Error in Polynomial2Pdf => %f",ret/(intg));
// if (ret/(intg)<0) printf("Error in Polynomial2Pdf => %f",ret/(intg));
//    return ret/(intg);
//    return ret*ret/(intg*intg);
    return ret*ret;
//    return ret;
}
 */
__device__ device_function_ptr ptr_to_Polynomial2       = device_Polynomial2;
__device__ device_function_ptr ptr_to_OffsetPolynomial2 = device_OffsetPolynomial2;
__device__ device_function_ptr ptr_to_MultiPolynomial2  = device_MultiPolynomial2;

// Constructor for single-variate Polynomial2, with optional zero point.
__host__
Polynomial2Pdf::Polynomial2Pdf(std::string n, Observable _x, std::vector<Variable> weights, unsigned int lowestDegree)
    : GooPdf(n, _x) {
    std::vector<unsigned int> pindices;
    pindices.push_back(lowestDegree);

    for(auto &weight : weights) {
        pindices.push_back(registerParameter(weight));
    }

    GET_FUNCTION_ADDR(ptr_to_Polynomial2);

    initialize(pindices);
}

// Constructor for single-variate Polynomial2, with optional zero point.
__host__ Polynomial2Pdf::Polynomial2Pdf(
    std::string n, Observable _x, std::vector<Variable> weights, Variable x0, unsigned int lowestDegree)
    : GooPdf(n, _x)
    , center(new Variable(x0)) {
    std::vector<unsigned int> pindices;
    pindices.push_back(lowestDegree);

    for(auto &weight : weights) {
        pindices.push_back(registerParameter(weight));
    }

    pindices.push_back(registerParameter(x0));
    GET_FUNCTION_ADDR(ptr_to_OffsetPolynomial2);

    initialize(pindices);
}

// Constructor for multivariate Polynomial2.
__host__ Polynomial2Pdf::Polynomial2Pdf(std::string n,
                                      std::vector<Observable> obses,
                                      std::vector<Variable> coeffs,
                                      std::vector<Variable> offsets,
                                      unsigned int maxDegree)
    : GooPdf(n) {
    unsigned int numParameters = 1;

    // For 1 observable, equal to n = maxDegree + 1.
    // For two, n*(n+1)/2, ie triangular number. This generalises:
    // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
    // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
    // ...
    for(unsigned int i = 0; i < obses.size(); ++i) {
        registerObservable(obses[i]);
        numParameters *= (maxDegree + 1 + i);
    }

    for(int i = observables.size(); i > 1; --i)
        numParameters /= i;

    while(numParameters > coeffs.size()) {
        char varName[100];
        sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));

        coeffs.emplace_back(varName, 0.1,-1000.,1000.);

        std::cout << "Warning: " << getName() << " created dummy variable " << varName
                  << "  to account for all terms.\n";
    }

    while(offsets.size() < obses.size()) {
        char varName[100];
        sprintf(varName, "%s_extra_offset_%i", getName().c_str(), static_cast<int>(offsets.size()));
        offsets.emplace_back(varName, 0);
    }

    std::vector<unsigned int> pindices;
    pindices.push_back(maxDegree);

    for(auto &offset : offsets) {
        pindices.push_back(registerParameter(offset));
    }

    for(auto &coeff : coeffs) {
        pindices.push_back(registerParameter(coeff));
    }

    GET_FUNCTION_ADDR(ptr_to_MultiPolynomial2);
    initialize(pindices);
}

__host__ fptype Polynomial2Pdf::integrate(fptype lo, fptype hi) const {
//     // This is *still* wrong. (13 Feb 2013.)
//      unsigned int *indices = host_indices + parameters;
//      fptype lowestDegree   = indices[1];
//  
//      if(center) {
// 	 hi -= host_params[indices[indices[0]]];
// 	 lo -= host_params[indices[indices[0]]];
//      }
//  
//      fptype ret = 0;
//  
//      for(int i = 2; i < indices[0] + (center ? 0 : 1); ++i) {
// 	 fptype powerPlusOne = lowestDegree + i - 2;
// 	 fptype curr	     = pow(hi, powerPlusOne);
// 	 curr -= pow(lo, powerPlusOne);
// 	 curr /= powerPlusOne;
// 	 ret += host_params[indices[i]] * curr;
//      }
//  
//      return ret;
       return 1;
}

__host__ fptype Polynomial2Pdf::getCoefficient(int coef) const {
    // NB! This function only works for single Polynomial2s.
    if(1 != observables.size()) {
        std::cout << "Warning: getCoefficient method of Polynomial2Pdf not implemented for multi-dimensional "
                     "Polynomial2s. Returning zero, which is very likely wrong.\n";
        return 0;
    }

    unsigned int *indices = host_indices + parameters;

    // True function is, say, ax^2 + bx + c.
    // We express this as (a'x^2 + b'x + c')*N.
    // So to get the true coefficient, multiply the internal
    // one by the normalisation. (In non-PDF cases the normalisation
    // equals one, which gives the same result.)

    // Structure is nP lowestdegree c1 c2 c3 nO o1
    if(coef < indices[1])
        return 0; // Less than least power.

    if(coef > indices[1] + (indices[0] - 1))
        return 0; // Greater than max power.

    fptype norm = normalize();
    norm        = (1.0 / norm);

    fptype param = host_params[indices[2 + coef - indices[1]]];
    return norm * param;
}
} // namespace GooFit
