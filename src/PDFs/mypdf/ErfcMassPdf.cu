#include "hip/hip_runtime.h"
#include <goofit/PDFs/mypdf/ErfcMassPdf.h>

namespace GooFit {

__device__ fptype device_ErfcMass (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2 + indices[0]]]; 
  fptype p0 = p[indices[1]];
  fptype p1 = p[indices[2]];
  fptype p2 = p[indices[3]];
  fptype p3 = p[indices[4]];

  fptype ret = (p2+p3*x)*erfc(p0*(x-p1));

  if (ret<=0) {
       printf("ErfcMass <=0!!!: x = %f , p0 = %f p1 = %f p2 = %f p3 = %f\n",x,p0,p1,p2,p3);
       return 0.;
  }else{
   return ret; 
  } 
}

__device__ device_function_ptr ptr_to_ErfcMass = device_ErfcMass; 

__host__ ErfcMassPdf::ErfcMassPdf (std::string n, Observable _x, Variable p0, Variable p1, Variable p2, Variable p3) 
  : GooPdf(n, _x)
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(p0));
  pindices.push_back(registerParameter(p1));
  pindices.push_back(registerParameter(p2));
  pindices.push_back(registerParameter(p3));
  GET_FUNCTION_ADDR(ptr_to_ErfcMass);
  initialize(pindices); 
}

} // namespace GooFit

