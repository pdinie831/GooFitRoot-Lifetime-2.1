#include "hip/hip_runtime.h"
#include "ExpGausPEEPdf.h"

EXEC_TARGET fptype device_ExpGausPEE (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2+indices[0]]]; 
  fptype sigma = evt[indices[3+indices[0]]]; 
  fptype mean  = p[indices[1]];
  fptype tau   = p[indices[2]];
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
  fptype ret = 0.5*tau; 
    fptype exparg = ret * (2*mean + tau*sigma*sigma - 2*x);
    fptype erfarg = (mean + tau*sigma*sigma - x) / (sigma * 1.4142135623);
//fptype exparg = tau * (tau*sigma*sigma/2. + mean-x);
//fptype erfarg = ((mean-x)/(sigma*sigma) + tau)*sigma /  1.4142135623;
//
  ret *= EXP(exparg); 
//  ret *= (1- ERF(erfarg)); 
  ret *= ERFC(erfarg); 
  if (ret<=0){
    printf("Error: ExpGausPEEPdf<=0!!! ==> x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,tau , ret);
  }   
//
    fptype lo = 0.01;
    fptype hi = 0.1 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - hi))*
                   erfc((tau+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - lo))*
                   erfc((tau+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  
		    
//  
//   fptype u_hi = tau * (hi - mean);
//   fptype v_hi = tau * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = tau * (lo - mean);
//   fptype v_lo = tau * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
  fptype integral = fabs(intg_hi-intg_lo);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = ret/integral;
 
//if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("\n\nExpGausPEEPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral =%f\n", x, sigma, mean,tau , ret, integral);
//  printf("ExpGausPEEPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral1=%f\n", x, sigma, mean,tau , ret, integral1);
//}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausPEEPdf->integrate  sigma=%f mean=%f tau=%f Integral=%f\n",    sigma, mean,tau , integral);
  printf("ExpGausPEEPdf->integrate  sigma=%f mean=%f tau=%f IntegralLO=%f\n",  sigma, mean,tau , intg_lo);
  printf("ExpGausPEEPdf->integrate  sigma=%f mean=%f tau=%f IntegralHI=%f\n",  sigma, mean,tau , intg_hi);
  printf("ExpGausPEEPdf->integrate  sigma=%f mean=%f tau=%f EXP(expa_hi)=%f\n",  sigma, mean,tau , EXP(expa_hi));
  printf("ExpGausPEEPdf->integrate  sigma=%f mean=%f tau=%f EXP(expa_lo)=%f\n",  sigma, mean,tau , EXP(expa_lo));
 }
 */// printf("ExpGausPEEPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausPEEPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausPEEPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral=%f\n", x, sigma, mean,tau , ret, integral);
//    printf("ExpGausPEEPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausPEEPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausPEEPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausPEEPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausPEEPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausPEEPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausPEEPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausPEEPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausPEEPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausPEEPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausPEEPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausPEEPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausPEEPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausPEEPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausPEEPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausPEE = device_ExpGausPEE; 

 __host__ ExpGausPEEPdf::ExpGausPEEPdf (std::string n, Observable* _x, Observable* _s,  Variable* mean, Variable* tau) 
  : GooPdf(_x, n)
{
//   registerObservable(_x); //already registered!!!
   registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
//  pindices.push_back(registerParameter(sigma));
  pindices.push_back(registerParameter(tau));
  GET_FUNCTION_ADDR(ptr_to_ExpGausPEE);
  initialise(pindices); 
}

 __host__ fptype ExpGausPEEPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype tau  = host_params[indices[2]]  ;
 fptype u_hi = tau * (hi - mean);
 fptype v_hi = tau * sigmaM;
// fptype v_hi = tau * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = tau * (lo - mean);
 fptype v_lo = tau * sigmaM;
// fptype v_lo = tau * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausPEEPdf->integrate  sigma=%f mean=%f tau=%f Integral=%f\n",  sigmaM, mean,tau , (intg_hi-intg_lo));
// printf("ExpGausPEEPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausPEEPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
 }
