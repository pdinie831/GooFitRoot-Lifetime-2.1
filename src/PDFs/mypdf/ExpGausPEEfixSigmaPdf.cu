#include "hip/hip_runtime.h"
#include "ExpGausPEEfixSigmaPdf.h"

EXEC_TARGET fptype device_ExpGausPEEfixSigma (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2+indices[0]]]; 
//  fptype sigma = evt[indices[3+indices[0]]]; 
  fptype sigma = p[indices[1]]; 
  fptype mean  = p[indices[2]];
  fptype tau   = p[indices[3]];
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
  fptype ret = 0.5*tau; 
//  fptype exparg = ret * (2*mean + tau*sigma*sigma - 2*x);
//  fptype erfarg = (mean + tau*sigma*sigma - x) / (sigma * 1.4142135623);
  fptype exparg = tau * (tau*sigma*sigma/2. + mean-x);
  fptype erfarg = ((mean-x)/(sigma*sigma) + tau)*sigma /  1.4142135623;
//
  ret *= EXP(exparg); 
//  ret *= (1- ERF(erfarg)); 
  ret *= ERFC(erfarg); 
  if (ret<=0){
    printf("Error: ExpGausPEEfixSigmaPdf<=0!!! ==> x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,tau , ret);
  }   
//
    fptype lo = 0.01;
    fptype hi = 0.1 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - hi))*
                   erfc((tau+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - lo))*
                   erfc((tau+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  
		    
//  
//   fptype u_hi = tau * (hi - mean);
//   fptype v_hi = tau * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = tau * (lo - mean);
//   fptype v_lo = tau * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
  fptype integral = fabs(intg_hi-intg_lo);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = ret/(integral);
 
if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("\n\nExpGausPEEfixSigmaPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral =%f\n", x, sigma, mean,tau , ret, integral);
//  printf("ExpGausPEEfixSigmaPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral1=%f\n", x, sigma, mean,tau , ret, integral1);
}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausPEEfixSigmaPdf->integrate  sigma=%f mean=%f tau=%f Integral=%f\n",    sigma, mean,tau , integral);
  printf("ExpGausPEEfixSigmaPdf->integrate  sigma=%f mean=%f tau=%f IntegralLO=%f\n",  sigma, mean,tau , intg_lo);
  printf("ExpGausPEEfixSigmaPdf->integrate  sigma=%f mean=%f tau=%f IntegralHI=%f\n",  sigma, mean,tau , intg_hi);
  printf("ExpGausPEEfixSigmaPdf->integrate  sigma=%f mean=%f tau=%f EXP(expa_hi)=%f\n",  sigma, mean,tau , EXP(expa_hi));
  printf("ExpGausPEEfixSigmaPdf->integrate  sigma=%f mean=%f tau=%f EXP(expa_lo)=%f\n",  sigma, mean,tau , EXP(expa_lo));
 }
 */// printf("ExpGausPEEfixSigmaPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausPEEfixSigmaPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausPEEfixSigmaPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral=%f\n", x, sigma, mean,tau , ret, integral);
//    printf("ExpGausPEEfixSigmaPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausPEEfixSigmaPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausPEEfixSigmaPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausPEEfixSigmaPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausPEEfixSigmaPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausPEEfixSigmaPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausPEEfixSigmaPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausPEEfixSigmaPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausPEEfixSigmaPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausPEEfixSigmaPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausPEEfixSigmaPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausPEEfixSigmaPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausPEEfixSigmaPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausPEEfixSigmaPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausPEEfixSigmaPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausPEEfixSigma = device_ExpGausPEEfixSigma; 

 __host__ ExpGausPEEfixSigmaPdf::ExpGausPEEfixSigmaPdf (std::string n, Observable* _x, Observable* _s,  Variable* mean, Variable* tau) 
  : GooPdf(_x, n)
{
//   registerObservable(_x); //already registered!!!
//   registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(_s));
  pindices.push_back(registerParameter(mean));
//  pindices.push_back(registerParameter(sigma));
  pindices.push_back(registerParameter(tau));
  GET_FUNCTION_ADDR(ptr_to_ExpGausPEEfixSigma);
  initialise(pindices); 
}

 __host__ fptype ExpGausPEEfixSigmaPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype tau  = host_params[indices[2]]  ;
 fptype u_hi = tau * (hi - mean);
 fptype v_hi = tau * sigmaM;
// fptype v_hi = tau * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = tau * (lo - mean);
 fptype v_lo = tau * sigmaM;
// fptype v_lo = tau * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausPEEfixSigmaPdf->integrate  sigma=%f mean=%f tau=%f Integral=%f\n",  sigmaM, mean,tau , (intg_hi-intg_lo));
// printf("ExpGausPEEfixSigmaPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausPEEfixSigmaPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
 }
