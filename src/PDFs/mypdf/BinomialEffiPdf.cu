#include "hip/hip_runtime.h"
#include <goofit/PDFs/mypdf/BinomialEffiPdf.h>

namespace GooFit {

__device__ fptype device_BinomialEffi(fptype *evt, fptype *p, unsigned int *indices) {
    fptype xRec     = evt[RO_CACHE(indices[2 + RO_CACHE(indices[0])])];
    fptype xGen     = evt[RO_CACHE(indices[2 + RO_CACHE(indices[0]) + 1])];

    fptype mean  = RO_CACHE(p[RO_CACHE(indices[1])]);
    fptype sigma = RO_CACHE(p[RO_CACHE(indices[2])]);

    fptype ret = exp(-0.5 * (x - mean) * (x - mean) / (sigma * sigma));

    return ret;
}

__device__ device_function_ptr ptr_to_BinomialEffi = device_BinomialEffi;

__host__ BinomialEffiPdf::BinomialEffiPdf(std::string n, Observable _r, Observable _g, Variable mean, Variable sigma)
    : GooPdf(n) {
    std::vector<unsigned int> pindices;
    pindices.push_back(registerObservable(_r));
    pindices.push_back(registerObservable(_g));
    pindices.push_back(registerParameter(mean));
    pindices.push_back(registerParameter(mean));
    pindices.push_back(registerParameter(sigma));
    GET_FUNCTION_ADDR(ptr_to_BinomialEffi);
    initialize(pindices);
}

__host__ fptype BinomialEffiPdf::integrate(fptype lo, fptype hi) const {
    return 1.0;
}

} // namespace GooFit
