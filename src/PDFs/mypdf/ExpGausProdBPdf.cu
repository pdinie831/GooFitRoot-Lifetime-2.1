#include "hip/hip_runtime.h"
#include <goofit/PDFs/mypdf/ExpGausProdBPdf.h>

namespace GooFit {

__device__ fptype device_ExpGausProdB (fptype *evt, fptype *p, unsigned int *indices) {
  fptype x         = evt[indices[2+indices[0]]]; 
  fptype sigma     = evt[indices[3+indices[0]]]; 
  fptype mean      = p[indices[1]];
  fptype lambda    = p[indices[2]];
  fptype sigmas    = p[indices[3]];
  fptype means     = p[indices[4]];
  fptype lambdas   = p[indices[5]];
  fptype lo        = p[indices[6]];
  fptype hi        = p[indices[7]];
  fptype los       = p[indices[8]];
  fptype his       = p[indices[9]];
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
    fptype ret = 0.5*lambda; 
    fptype exparg = ret * (2*mean + lambda*sigma*sigma - 2*x);
    fptype erfarg = (mean + lambda*sigma*sigma - x) / (sigma * 1.4142135623);
    ret *= exp(exparg); 
    ret *= erfc(erfarg); 

  if (ret<=0){
    printf("Error: ExpGausProdBPdf<=0!!! ==> x=%f  sigma=%f mean=%f lambda=%f  ret=%f lo=%f  hi=%f \n", 
    x, sigma,mean,lambda, ret, lo,hi);
  }   
    fptype y = sigma;
    fptype rets = 0.5*lambdas; 
    fptype expargs = rets * (2*means + lambdas*sigmas*sigmas - 2*y);
    fptype erfargs = (means + lambdas*sigmas*sigmas - y) / (sigmas * 1.4142135623);
//fptype exparg = lambda * (lambda*sigma*sigma/2. + mean-x);
//fptype erfarg = ((mean-x)/(sigma*sigma) + lambda)*sigma /  1.4142135623;
//
    rets *= exp(expargs); 
    rets *= erfc(erfargs); 
    ret=rets*ret;
  if (rets<=0){
    printf("Error: ExpGausProdBPdf<=0!!! ==> x=%f  sigmas=%f means=%f lambdas=%f rets=%f los=%f his=%f\n", 
    x,  sigmas, means,lambdas ,rets,los,his);
  }   
//
//    fptype lo = 0.01;
//    fptype hi = 0.1 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-exp(lambda*(lambda*sigma*sigma/2.+mean - hi))*
                   erfc((lambda+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-exp(lambda*(lambda*sigma*sigma/2.+mean - lo))*
                   erfc((lambda+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  

//  fptype los = 0.0003;
//  fptype his = 0.007 ;
  fptype intg_his = 0.5*(1-exp(lambdas*(lambdas*sigmas*sigmas/2.+means - his))*
                   erfc((lambdas+(means-his)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((his-means)/(sigmas*1.4142135623)));   
  fptype intg_los = 0.5*(1-exp(lambdas*(lambdas*sigmas*sigmas/2.+means - los))*
                   erfc((lambdas+(means-los)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((los-means)/(sigmas*1.4142135623)));  
		    
//  
//   fptype u_hi = lambda * (hi - mean);
//   fptype v_hi = lambda * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = lambda * (lo - mean);
//   fptype v_lo = lambda * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
   fptype integral = fabs(intg_hi-intg_lo)*fabs(intg_his-intg_los);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = ret/integral;
 
//if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("\n\nExpGausProdBPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral =%f\n", x, sigma, mean,lambda , ret, integral);
//  printf("ExpGausProdBPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral1=%f\n", x, sigma, mean,lambda , ret, integral1);
//}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausProdBPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",    sigma, mean,lambda , integral);
  printf("ExpGausProdBPdf->integrate  sigma=%f mean=%f lambda=%f IntegralLO=%f\n",  sigma, mean,lambda , intg_lo);
  printf("ExpGausProdBPdf->integrate  sigma=%f mean=%f lambda=%f IntegralHI=%f\n",  sigma, mean,lambda , intg_hi);
  printf("ExpGausProdBPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_hi)=%f\n",  sigma, mean,lambda , EXP(expa_hi));
  printf("ExpGausProdBPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_lo)=%f\n",  sigma, mean,lambda , EXP(expa_lo));
 }
 */// printf("ExpGausProdBPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdBPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausProdBPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral=%f\n", x, sigma, mean,lambda , ret, integral);
//    printf("ExpGausProdBPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausProdBPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausProdBPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausProdBPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausProdBPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausProdBPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausProdBPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausProdBPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausProdBPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausProdBPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausProdBPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausProdBPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausProdBPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausProdBPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausProdBPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

__device__ device_function_ptr ptr_to_ExpGausProdB = device_ExpGausProdB; 

 __host__ ExpGausProdBPdf::ExpGausProdBPdf (std::string n, Observable _x, Observable _s,  Variable mean, Variable lambda, Variable sigmas, Variable means, Variable lambdas,  Variable lo, Variable hi, Variable los, Variable his) 
  : GooPdf(n, _x)
{
//   registerObservable(_x); //already registered!!!
  registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(lambda));
  pindices.push_back(registerParameter(sigmas));
  pindices.push_back(registerParameter(means));
  pindices.push_back(registerParameter(lambdas));
  pindices.push_back(registerParameter(lo));
  pindices.push_back(registerParameter(hi));
  pindices.push_back(registerParameter(los));
  pindices.push_back(registerParameter(his));
  GET_FUNCTION_ADDR(ptr_to_ExpGausProdB);
  initialize(pindices); 
}

 __host__ fptype ExpGausProdBPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype lambda  = host_params[indices[2]]  ;
 fptype u_hi = lambda * (hi - mean);
 fptype v_hi = lambda * sigmaM;
// fptype v_hi = lambda * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = lambda * (lo - mean);
 fptype v_lo = lambda * sigmaM;
// fptype v_lo = lambda * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausProdBPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",  sigmaM, mean,lambda , (intg_hi-intg_lo));
// printf("ExpGausProdBPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdBPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
}
 
} // namespace GooFit
