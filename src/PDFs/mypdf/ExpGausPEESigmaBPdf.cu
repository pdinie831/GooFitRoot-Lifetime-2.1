#include "hip/hip_runtime.h"
#include "ExpGausPEESigmaBPdf.h"

EXEC_TARGET fptype device_ExpGausPEESigmaB (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2+indices[0]]]; 
//  fptype sigma = evt[indices[3+indices[0]]]; 
  fptype sigma = p[indices[1]]; 
  fptype mean  = p[indices[2]];
  fptype tau   = p[indices[3]];
  fptype lo    = p[indices[4]];
  fptype hi    = p[indices[5]];
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
  fptype ret = 0.5*tau; 
//  fptype exparg = ret * (2*mean + tau*sigma*sigma - 2*x);
//  fptype erfarg = (mean + tau*sigma*sigma - x) / (sigma * 1.4142135623);
  fptype exparg = tau * (tau*sigma*sigma/2. + mean-x);
  fptype erfarg = ((mean-x)/(sigma*sigma) + tau)*sigma /  1.4142135623;
//
  ret *= EXP(exparg); 
//  ret *= (1- ERF(erfarg)); 
  ret *= ERFC(erfarg); 
  if (ret<=0){
    printf("Error: ExpGausPEESigmaBPdf<=0!!! ==> x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,tau , ret);
  }   
//
//    fptype lo = 0.0003;
//    fptype hi = 0.007 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - hi))*
                   erfc((tau+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - lo))*
                   erfc((tau+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  
		    
//  
//   fptype u_hi = tau * (hi - mean);
//   fptype v_hi = tau * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = tau * (lo - mean);
//   fptype v_lo = tau * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
  fptype integral = fabs(intg_hi-intg_lo);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = ret/(integral);
 
//if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("\n\nExpGausPEESigmaBPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral =%f\n", x, sigma, mean,tau , ret, integral);
//  printf("ExpGausPEESigmaBPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral1=%f\n", x, sigma, mean,tau , ret, integral1);
//}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausPEESigmaBPdf->integrate  sigma=%f mean=%f tau=%f Integral=%f\n",    sigma, mean,tau , integral);
  printf("ExpGausPEESigmaBPdf->integrate  sigma=%f mean=%f tau=%f IntegralLO=%f\n",  sigma, mean,tau , intg_lo);
  printf("ExpGausPEESigmaBPdf->integrate  sigma=%f mean=%f tau=%f IntegralHI=%f\n",  sigma, mean,tau , intg_hi);
  printf("ExpGausPEESigmaBPdf->integrate  sigma=%f mean=%f tau=%f EXP(expa_hi)=%f\n",  sigma, mean,tau , EXP(expa_hi));
  printf("ExpGausPEESigmaBPdf->integrate  sigma=%f mean=%f tau=%f EXP(expa_lo)=%f\n",  sigma, mean,tau , EXP(expa_lo));
 }
 */// printf("ExpGausPEESigmaBPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausPEESigmaBPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausPEESigmaBPdf x=%f  sigma=%f mean=%f tau=%f ret=%f integral=%f\n", x, sigma, mean,tau , ret, integral);
//    printf("ExpGausPEESigmaBPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausPEESigmaBPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausPEESigmaBPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausPEESigmaBPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausPEESigmaBPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausPEESigmaBPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausPEESigmaBPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausPEESigmaBPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausPEESigmaBPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausPEESigmaBPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausPEESigmaBPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausPEESigmaBPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausPEESigmaBPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausPEESigmaBPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausPEESigmaBPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausPEESigmaB = device_ExpGausPEESigmaB; 

 __host__ ExpGausPEESigmaBPdf::ExpGausPEESigmaBPdf (std::string n, Observable* _x, Observable* _s,  Variable* mean, Variable* tau, Variable* lo, Variable* hi  ) 
  : GooPdf(_x, n)
{
//   registerObservable(_x); //already registered!!!
//   registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(_s));
  pindices.push_back(registerParameter(mean));
//  pindices.push_back(registerParameter(sigma));
  pindices.push_back(registerParameter(tau));
  pindices.push_back(registerParameter(lo));
  pindices.push_back(registerParameter(hi));
  GET_FUNCTION_ADDR(ptr_to_ExpGausPEESigmaB);
  initialise(pindices); 
}

 __host__ fptype ExpGausPEESigmaBPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype tau  = host_params[indices[2]]  ;
 fptype u_hi = tau * (hi - mean);
 fptype v_hi = tau * sigmaM;
// fptype v_hi = tau * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = tau * (lo - mean);
 fptype v_lo = tau * sigmaM;
// fptype v_lo = tau * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausPEESigmaBPdf->integrate  sigma=%f mean=%f tau=%f Integral=%f\n",  sigmaM, mean,tau , (intg_hi-intg_lo));
// printf("ExpGausPEESigmaBPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausPEESigmaBPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
 }
