#include "hip/hip_runtime.h"
#include "ExpGausProdEffiBPdf.h"

EXEC_TARGET fptype device_ExpGausProdEffiB (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x         = evt[indices[2+indices[0]]]; 
  fptype sigma     = evt[indices[3+indices[0]]]; 
  fptype mean      = p[indices[1]];
  fptype lambda    = p[indices[2]];
  fptype sigmas    = p[indices[3]];
  fptype means     = p[indices[4]];
  fptype lambdas   = p[indices[5]];
  fptype lo        = p[indices[6]];
  fptype hi        = p[indices[7]];
  fptype los       = p[indices[8]];
  fptype his       = p[indices[9]];
  
  fptype p0 =	1.04436e-01;
  fptype p1 =  -1.32602e-01;
  fptype p2 =	3.45532e-01;
  fptype p3 =  -6.40565e-01;
 
  fptype Effi = p0 + p1*x + p2*x*x+ p3*x*x*x ;
  fptype IEffihi = p0*hi + p1*hi*hi + p2*hi*hi*hi+ p3*hi*hi*hi*hi ;
  fptype IEffilo = p0*lo + p1*lo*lo + p2*lo*lo*lo+ p3*lo*lo*lo*lo ;
  fptype IEffi = IEffihi-IEffilo;
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
    fptype ret = 0.5*lambda; 
    fptype exparg = ret * (2*mean + lambda*sigma*sigma - 2*x);
    fptype erfarg = (mean + lambda*sigma*sigma - x) / (sigma * 1.4142135623);
    ret *= EXP(exparg); 
    ret *= ERFC(erfarg); 

    fptype y = sigma;
    fptype rets = 0.5*lambdas; 
    fptype expargs = rets * (2*means + lambdas*sigmas*sigmas - 2*y);
    fptype erfargs = (means + lambdas*sigmas*sigmas - y) / (sigmas * 1.4142135623);
//fptype exparg = lambda * (lambda*sigma*sigma/2. + mean-x);
//fptype erfarg = ((mean-x)/(sigma*sigma) + lambda)*sigma /  1.4142135623;
//
    rets *= EXP(expargs); 
    rets *= ERFC(erfargs); 
    ret=rets*ret;
  if (ret<=0){
    printf("Error: ExpGausProdEffiBPdf<=0!!! ==> x=%f  sigma=%f mean=%f lambda=%f sigmas=%f means=%f lambdas=%fret=%f\n", x, sigma, mean,lambda , sigmas, means,lambdas ,ret);
    printf("Error: ExpGausProdEffiBPdf<=0!!! ==> lo=%f  hi=%f los=%f his=%f \n", lo,hi,los,his);
  }   
//
//    fptype lo = 0.01;
//    fptype hi = 0.1 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-EXP(lambda*(lambda*sigma*sigma/2.+mean - hi))*
                   erfc((lambda+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(lambda*(lambda*sigma*sigma/2.+mean - lo))*
                   erfc((lambda+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  

//  fptype los = 0.0003;
//  fptype his = 0.007 ;
  fptype intg_his = 0.5*(1-EXP(lambdas*(lambdas*sigmas*sigmas/2.+means - his))*
                   erfc((lambdas+(means-his)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((his-means)/(sigmas*1.4142135623)));   
  fptype intg_los = 0.5*(1-EXP(lambdas*(lambdas*sigmas*sigmas/2.+means - los))*
                   erfc((lambdas+(means-los)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((los-means)/(sigmas*1.4142135623)));  
		    
//  
//   fptype u_hi = lambda * (hi - mean);
//   fptype v_hi = lambda * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = lambda * (lo - mean);
//   fptype v_lo = lambda * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
   fptype integral = fabs(intg_hi-intg_lo)*fabs(intg_his-intg_los);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = Effi/IEffi*ret/integral;
 
//if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("\n\nExpGausProdEffiBPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral =%f\n", x, sigma, mean,lambda , ret, integral);
//  printf("ExpGausProdEffiBPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral1=%f\n", x, sigma, mean,lambda , ret, integral1);
//}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausProdEffiBPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",    sigma, mean,lambda , integral);
  printf("ExpGausProdEffiBPdf->integrate  sigma=%f mean=%f lambda=%f IntegralLO=%f\n",  sigma, mean,lambda , intg_lo);
  printf("ExpGausProdEffiBPdf->integrate  sigma=%f mean=%f lambda=%f IntegralHI=%f\n",  sigma, mean,lambda , intg_hi);
  printf("ExpGausProdEffiBPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_hi)=%f\n",  sigma, mean,lambda , EXP(expa_hi));
  printf("ExpGausProdEffiBPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_lo)=%f\n",  sigma, mean,lambda , EXP(expa_lo));
 }
 */// printf("ExpGausProdEffiBPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdEffiBPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausProdEffiBPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral=%f\n", x, sigma, mean,lambda , ret, integral);
//    printf("ExpGausProdEffiBPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausProdEffiBPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausProdEffiBPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausProdEffiBPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausProdEffiBPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausProdEffiBPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausProdEffiBPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausProdEffiBPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausProdEffiBPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausProdEffiBPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausProdEffiBPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausProdEffiBPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausProdEffiBPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausProdEffiBPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausProdEffiBPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausProdEffiB = device_ExpGausProdEffiB; 

 __host__ ExpGausProdEffiBPdf::ExpGausProdEffiBPdf (std::string n, Observale* _x, Observable* _s,  Variable* mean, Variable* lambda, Variable* sigmas, Variable* means, Variable* lambdas, 
                                             Variable* lo, Variable* hi, Variable* los, Variable* his) 
  : GooPdf(_x, n)
{
//   registerObservable(_x); //already registered!!!
   registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(lambda));
  pindices.push_back(registerParameter(sigmas));
  pindices.push_back(registerParameter(means));
  pindices.push_back(registerParameter(lambdas));
  pindices.push_back(registerParameter(lo));
  pindices.push_back(registerParameter(hi));
  pindices.push_back(registerParameter(los));
  pindices.push_back(registerParameter(his));
  GET_FUNCTION_ADDR(ptr_to_ExpGausProdEffiB);
  initialise(pindices); 
}

 __host__ fptype ExpGausProdEffiBPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype lambda  = host_params[indices[2]]  ;
 fptype u_hi = lambda * (hi - mean);
 fptype v_hi = lambda * sigmaM;
// fptype v_hi = lambda * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = lambda * (lo - mean);
 fptype v_lo = lambda * sigmaM;
// fptype v_lo = lambda * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausProdEffiBPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",  sigmaM, mean,lambda , (intg_hi-intg_lo));
// printf("ExpGausProdEffiBPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdEffiBPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
 }
