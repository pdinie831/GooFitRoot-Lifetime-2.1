#include "hip/hip_runtime.h"
#include "TrivarGaussianConstrPdf.h"

EXEC_TARGET fptype device_TrivarGaussianConstr (fptype* evt, fptype* p, unsigned int* indices) {
//   fptype x = evt[indices[2 + indices[0]]]; 
//   fptype y = evt[indices[3 + indices[0]]]; 
  fptype x      = p[indices[1]];
  fptype y      = p[indices[2]];
  fptype z      = p[indices[3]];
  fptype mean1  = p[indices[4]];
  fptype mean2  = p[indices[5]];
  fptype mean3  = p[indices[6]];
  fptype sigma11 = p[indices[7]];
  fptype sigma22 = p[indices[8]];
  fptype sigma33 = p[indices[9]];
  fptype sigma12 = p[indices[10]];
  fptype sigma13 = p[indices[11]];
  fptype sigma23 = p[indices[12]];
  
//  const fptype pi = 3.14159265359;
  
  fptype dx      = x-mean1;
  fptype dy      = y-mean2;
  fptype dz      = x-mean3;
  
  fptype a11 = sigma33 * sigma22 - sigma23 * sigma23  ;
  fptype a12 = sigma13 * sigma23 - sigma33 * sigma12  ;
  fptype a13 = sigma12 * sigma23 - sigma13 * sigma22  ;

  fptype a22 = sigma33 * sigma11 - sigma13 * sigma13  ;
  fptype a23 = sigma12 * sigma13 - sigma11 * sigma23  ;
					       
  fptype a33 = sigma11 * sigma22 - sigma12 * sigma12  ;
  
  fptype Det  = (sigma11 * a11) + (sigma12 * a12) + (sigma13 * a13);
  
  fptype ret = (dx*dx*a11+dy*dy*a22+dz*dz*a33+2*dx*dy*a12+2*dx*dz*a13+2*y*z*a23)/Det;
  
  if (Det <=0) {
     printf("Error: Det<=0.!!! ==> x=%f  y=%f z=%f sigma11=%f sigma22=%f sigma33=%f sigma12=%f sigma13=%f sigma23=%f \n",x,y,z,sigma11,sigma22,sigma33,sigma12,sigma13,sigma23);
     return 0;
   } 

//  return -0.5*ret; 
  return EXP(-0.5*ret); 
}

MEM_DEVICE device_function_ptr ptr_to_TrivarGaussianConstr = device_TrivarGaussianConstr; 

__host__ TrivarGaussianConstrPdf::TrivarGaussianConstrPdf 
(std::string n, Variable* _xdummy, Observable* _x, Observable* _y, Observable* _z, 
                                   Variable* mean1, Variable* mean2, Variable* mean3, 
				   Variable* sigma11, Variable* sigma22, Variable* sigma33,
				   Variable* sigma12, Variable* sigma13, Variable* sigma23) 
  : GooPdf(_xdummy, n) 
{
//  registerObservable(_x);
//  registerObservable(_y);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(_x));
  pindices.push_back(registerParameter(_y));
  pindices.push_back(registerParameter(_z));
  pindices.push_back(registerParameter(mean1));
  pindices.push_back(registerParameter(mean2));
  pindices.push_back(registerParameter(mean3));
  pindices.push_back(registerParameter(sigma11));
  pindices.push_back(registerParameter(sigma22));
  pindices.push_back(registerParameter(sigma33));
  pindices.push_back(registerParameter(sigma12));
  pindices.push_back(registerParameter(sigma13));
  pindices.push_back(registerParameter(sigma23));
  GET_FUNCTION_ADDR(ptr_to_TrivarGaussianConstr);
  initialise(pindices); 
}

 __host__ fptype TrivarGaussianConstrPdf::integrate (fptype lo, fptype hi) const {
//   //static const fptype root2 = sqrt(2.);
//   static const fptype rootPi = sqrt(atan2(0.0,-1.0));
//   static const fptype rootPiBy2 = rootPi / root2;
//   
//   unsigned int* indices = host_indices+parameters; 
//   fptype xscale = root2*host_params[indices[2]];
// 
//   /*
//   std::cout << "TrivarGaussianConstr integral: " 
// 	    << xscale << " "
// 	    << host_params[indices[1]] << " "
// 	    << host_params[indices[2]] << " "
// 	    << ERF((hi-host_params[indices[1]])/xscale) << " "
// 	    << ERF((lo-host_params[indices[1]])/xscale) << " "
// 	    << rootPiBy2*host_params[indices[2]]*(ERF((hi-host_params[indices[1]])/xscale) -
// 						  ERF((lo-host_params[indices[1]])/xscale)) 
// 	    << std::endl; 
//   */
//   return rootPiBy2*host_params[indices[2]]*(ERF((hi-host_params[indices[1]])/xscale) - 
//   					    ERF((lo-host_params[indices[1]])/xscale));

  // Integral over all R. 
//   fptype sigma = host_params[indices[2]];
//   sigma *= root2*rootPi;
//   return sigma; 
return 1.;
}

