#include "hip/hip_runtime.h"
#define MAX_THREADS_PER_BLOCK 512
#define MIN_BLOCKS_PER_MP     20
#include <goofit/PDFs/mypdf/FastBernsteinPdf.h>
#include <goofit/Variable.h>

//   __global__ void
//   __launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)

namespace GooFit {

/* a struct for storing double numbers */
// struct bernVal {
//   double bernFunc;
//   double bernIntg;
// };


__forceinline__ __device__ fptype device_coeffbinomial(fptype enne, fptype kappa){
 
        fptype factor=1.;
        for(fptype i = 1; i <=kappa; ++i) {
          factor *= (enne+1-i)/i; 
        }	 
 
        if (factor<=0 ){
	 printf("Error in FastBernsteinPdf coeffbinomial=> factor = %f enne=%f kappa=%f",factor,enne,kappa);
         return 0;
	} 
       return factor;
}
__forceinline__ __device__ fptype  device_bernsteinkn_func(fptype x, fptype enne, fptype kappa){
 
   return device_coeffbinomial(enne,kappa)*pow(x,kappa)*pow(1.0-x,enne-kappa);


}
/*  __device__ fptype  device_fastbernsteinkn_intg(fptype x, fptype enne, fptype kappa){
 
//  	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
//       printf("==================================================\n");
//       printf("==================================================\n");
//      }
//       struct bernVal results;
//       results.bernFunc = 0;
//       results.bernIntg = 0;
      if (x<0 || x>1 ){
       printf(" Error in bernsteinkn_intg  x=%5.15f out of range [0,1]\n",x);
       return 0.;
      }
//      if (kappa>enne) return 0;
//      bernkn *= pow(x,kappa) ;
//      bernkn *= pow(1.0-x,enne-kappa) ;
      fptype integbernkn = 0;
      fptype ifactni = 0;
      fptype ifactik = 0;
      
       for(fptype i = kappa; i <=enne ; ++i) {
// n!/(i!(n-i)!)
//         ifactni=1;
//         for(float l = 1; l <=i; ++l) {
//           ifactni *= (maxDegree+1-l)/l; 
//         }	 

        ifactni =  device_coeffbinomial(enne,i);
// i!/(k!(i-k)!)
//         ifactik=1;
//         for(float l = 1; l <=k; ++l) {
//           ifactik *= (i+1-l)/l; 
//         }	
 
        ifactik =  device_coeffbinomial(i,kappa);
//
//        bernkn      += ifactni*ifactik*pow(menuno,i-kappa)*pow(x, i) ;
        integbernkn += ifactni*ifactik*pow(-1.0,i-kappa)/(i+1);
//	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
//          printf("pow(x=%5.15f,i=%5.15f)=%5.15f\n",x,i,pow(x, i ));
//          printf("pow(-1=%5.15f,i-kappa)=%5.15f\n",menuno,pow(menuno,i-kappa));
//          printf("bernsteinkn=%5.15f integral=%5.15f ifactni=%5.15f ifactik=%5.15f \n",bernkn,integbernkn,ifactni,ifactik);
//          printf("bernsteinkn=%f integral=%f kappa=%f i=%f enne=%f ni=%f nk=%f\n",bernkn,integbernkn,kappa,i,enne,ifactni,ifactik);
//        }
       }

       if (integbernkn<=0 ){
//	if ((52 == THREADIDX) && (0 == BLOCKIDX)){
         printf(" Error in bernsteinkn_intg x=%5.15f integral = %5.15f THREADIDX=%d BLOCKIDX=%d\n", x,kappa,enne,integbernkn,THREADIDX,BLOCKIDX);
//	}
       }
//       results.bernFunc = bernkn;
//       results.bernIntg = integbernkn;
       return integbernkn;
}
 *///
//
//
//
__device__ fptype device_FastBernstein(fptype *evt, fptype *p, unsigned int *indices) {
    // Structure is nP lowestdegree c1 c2 c3 nO o1
     
     
//    struct bernVal bernknval;

    int numParams = (indices[0]) ;
    int maxDegree = (indices[1]);

    fptype x   = evt[(indices[2 + (indices[0])])];
    fptype ret = 0.0;
//    fptype integret = 0;
//    fptype bernkn = 0;
//    fptype integbernkn = 0;
    fptype intg_1 = 0.0;
    int    ipar = 2;
//    fptype ifactni=1;
//    fptype ifactik=1;
    fptype xmin=p[(indices[numParams-1])];
    fptype xmax=p[(indices[numParams])];
    x=(x-xmin)/(xmax-xmin);
//     printf("FastBernsteinPdf => limit xmin= %f xmax= %f\n",xmin,xmax);
//     return 0;


//     for(int i = 2; i < numParams; ++i) {
//         ret += (p[(indices[i])]) * pow(x, lowestDegree + i - 2);
//     }
    
      float k;
//      float i;
      for(k = 0; k <=maxDegree; ++k) {
       if (ipar>numParams-1){
        printf("Error in FastBernsteinPdf => ipar=%d > numParams=%d\n",ipar,numParams);
        return 0;
       }
       ret      += (p[(indices[ipar])]) * device_bernsteinkn_func(x,maxDegree,k);
       intg_1   += (p[(indices[ipar])]);
//       integret += (p[(indices[ipar])]) * device_bernsteinkn_intg(x,maxDegree,k);
//       printf("FastBernsteinPdf => %f integral = %f k=%d numparam=%d par=%f\n",ret,integret,k,numParams,(p[(indices[ipar])]));
       ipar++;
      }
      intg_1=(maxDegree+1)/intg_1/(xmax-xmin);
//       printf("FastBernsteinPdf => %f int = %f\n",ret,integret);
    if(ret<1.E-30)  return 1.E-30;
    return ret*intg_1;
//    return 0.;
}

/* __device__ fptype device_OffsetFastBernstein(fptype *evt, fptype *p, unsigned int *indices) {
    int numParams    = (indices[0]);
    int lowestDegree = (indices[1]);

    fptype x = evt[(indices[2 + numParams])];
    x -= (p[(indices[numParams])]);
    fptype ret = 0;

    for(int i = 2; i < numParams; ++i) {
        ret += (p[(indices[i])]) * pow(x, lowestDegree + i - 2);
    }

    return ret*ret;
}
*/
__device__ fptype device_MultiFastBernstein(fptype *evt, fptype *p, unsigned int *indices) {
//       if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//        printf("==================================================\n");
//       }
// //     struct bernVal bernknvalx;
//     struct bernVal bernknvaly;
//     struct bernVal bernknvalz;
    int numObservables = (indices[(indices[0]) + 1]);
    int maxDegree1      = (indices[1]);
    int maxDegree2      = (indices[2]);
    int maxDegree3      = (indices[3]);
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiFastBernstein 0=%d 0+1=%d 0+2=%d 0+3=%d\n",indices[(indices[0])],indices[(indices[0])+1],indices[(indices[0])+2],indices[(indices[0])+3]);
//      printf("MultiFastBernstein numObservables=%d maxDegree1=%d maxDegree2=%d maxDegree3=%d\n",numObservables,indices[1],indices[2],indices[3]);
//      }
    if (numObservables!=3) {
     printf("device_MultiFastBernstein error: Max Number of Observables is = 3!!! numObservables = %d\n",numObservables);
     return -100;
    }
 
    fptype x    = (evt[(indices[2 + (indices[0]) ])]); // x, y, z...
    fptype y    = (evt[(indices[2 + (indices[0]) + 1])]); // x, y, z...
    fptype z    = (evt[(indices[2 + (indices[0]) + 2])]); // x, y, z...
//      if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//      printf("MultiFastBernstein x=%5.15f y=%5.15f z=%5.15f %d %d %d\n",x,y,z,numObservables,indices[1],indices[2],indices[3]);
//      }
    fptype xmin = (p[(indices[4 ])]);
    fptype xdif = (p[(indices[5 ])])-(p[(indices[4 ])]);
    x=(x-xmin)/xdif;
    fptype ymin = (p[(indices[6])]);
    fptype ydif = (p[(indices[7])])-(p[(indices[6])]);
    y=(y-ymin)/ydif;
    fptype zmin = (p[(indices[8])]);
    fptype zdif = (p[(indices[9])])-(p[(indices[8])]);
    z=(z-zmin)/zdif;
    
//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	printf("MultiFastBernstein xmin=%5.15f xmax = %5.15f\n",xmin,xdif);
// 	printf("MultiFastBernstein ymin=%5.15f ymax = %5.15f\n",ymin,ydif); 
// 	printf("MultiFastBernstein zmin=%5.15f zmax = %5.15f\n",zmin,zdif);
// 	printf("MultiFastBernstein [0,1] x=%5.15f y=%5.15f z=%5.15f \n",x,y,z);
//        
//        }
       int ipar =4 + 2*numObservables;
//       int kk = 0;
//       int ii = 0;
//       int jj = 0;
       fptype func =0;
       fptype intg_1 =0;
       for(int i = 0; i <= maxDegree1 ; ++i) {
//       jj = 0;
         for(int j = 0; j <= maxDegree2 ; ++j) {
//	  std::cout<<"func = par["<<ipar<<"]*x^"<<kk<<"*y^"<<jj<<std::endl;
//          ii = 0;
          for(int k = 0; k <= maxDegree3 ; ++k) {
//	   std::cout<<"func = par["<<ipar<<"]*x^"<<ii<<"*y^"<<jj<<"*z^"<<kk<<std::endl;
    	   fptype bernknvalx =  device_coeffbinomial(maxDegree1,i)*pow(x,i)*pow(1.0-x,maxDegree1-i);
    	   fptype bernknvaly =  device_coeffbinomial(maxDegree2,j)*pow(y,j)*pow(1.0-y,maxDegree2-j);
    	   fptype bernknvalz =  device_coeffbinomial(maxDegree3,k)*pow(z,k)*pow(1.0-z,maxDegree3-k);
//     	   fptype bernknvalx =  device_bernsteinkn_func(x,maxDegree1,i);
//  	   fptype bernknvaly =  device_bernsteinkn_func(y,maxDegree2,j);
//  	   fptype bernknvalz =  device_bernsteinkn_func(z,maxDegree3,k);
//	   fptype bernknintx =  device_bernsteinkn_intg(x,maxDegree1,i);
//	   fptype bernkninty =  device_bernsteinkn_intg(y,maxDegree2,j);
//	   fptype bernknintz =  device_bernsteinkn_intg(z,maxDegree3,k);
//            func +=(p[(indices[ipar])])*bernknvalx*bernknvaly*bernknvalz;
//            intg +=(p[(indices[ipar])])*bernknintx*bernkninty*bernknintz;
           func   +=(p[(indices[ipar])])*bernknvalx*bernknvaly*bernknvalz;
           intg_1 +=(p[(indices[ipar])]);
// 	    if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  	     printf("MultiFastBernstein  par = %f       \n",(p[(indices[ipar])]));
// 	     printf("MultiFastBernstein  par = %f       B_(%d,%d,%d) = %f intg=%f\n",(p[(indices[ipar])]),ii,jj,kk,bernknvalx,bernknintx);
// 	    } 

//        if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	printf("MultiFastBernstein MaxDegree=%d coefficient = %f   number = %d\n",maxDegree,(p[(indices[ipar])]),ipar-2-2*numObservables);
//        } 
	   
	   ipar++;
//           ii = (jj+kk+ii<maxDegree?++ii:0);
	  }
//          jj = (jj+kk+ii<maxDegree?++jj:0);
	  
	 
         }
//         kk= (jj+kk+ii<maxDegree?++kk:0);
       }
//       return  func*func;
//       return  func/(intg);
//      return  func/(intg)/xdif/ydif/zdif;
//      return  func/(intg)/xdif/ydif/zdif;
      if(func<1.E-30)  return 1.E-30;
      intg_1 = (1.0+maxDegree1)*(1.0+maxDegree2)*(1.0+maxDegree3)/intg_1;
      intg_1 = intg_1/(xdif*ydif*zdif);
      fptype ret = func*intg_1;
//      if(ret>1.0) return 0.99999;
      return  ret;
 }
// __device__ fptype device_MultiFastBernstein(fptype *evt, fptype *p, unsigned int *indices) {
//     // Structure is nP, maxDegree, offset1, offset2, ..., coeff1, coeff2, ..., nO, o1, o2, ...
// 
//     struct bernVal bernknval;
//     int numObservables = (indices[(indices[0]) + 1]);
//     int maxDegree      = (indices[1])+1;
//  
// //     printf("MultiFastBernstein CosL xmin = %f   xmax = %f\n",(p[(indices[4 + 0])]),(p[(indices[4 + 0]+1)]));
// //     printf("MultiFastBernstein CosK xmin = %f   xmax = %f\n",(p[(indices[4 + 1])]),(p[(indices[4 + 1]+1)]));
// //     printf("MultiFastBernstein Phi  xmin = %f   xmax = %f\n",(p[(indices[4 + 2])]),(p[(indices[4 + 2]+1)]));
// //     return 0;
//     
// //    printf("MultiFastBernstein  maxDegree = %d   numObservables = %d\n",maxDegree,numObservables);
// //    return 0;
//     
//     // Only appears in construction (maxDegree + 1) or (x > maxDegree), so
//     // may as well add the one and use >= instead.
// 
//     // Technique is to iterate over the full n-dimensional box, skipping matrix elements
//     // whose sum of indices is greater than maxDegree. Notice that this is increasingly
//     // inefficient as n grows, since a larger proportion of boxes will be skipped.
//     int numBoxes = 1;
//     
//     for(int i = 0; i < numObservables; ++i)
//         numBoxes *= maxDegree;
// 
// //     int coeffNumber = 2 + numObservables; // Index of first coefficient is 2 + nO, not 1 + nO, due to maxDegree. (nO
// //                                           // comes from offsets.)
//     int coeffNumber = 2 + 2*numObservables-1; // Index of first coefficient is 2 + nO, not 1 + nO, due to maxDegree. (nO
//                                           // comes from offsets.)
// //    fptype ret = (p[(indices[coeffNumber++])]); // Coefficient of constant term.
// //         printf("MultiFastBernstein  coefficient = %f   number = %d\n",(p[(indices[coeffNumber])]),(indices[coeffNumber]));
// //         printf("MultiFastBernstein  coefficient = %f   number = %d\n",(p[(indices[coeffNumber]+85)]),(indices[coeffNumber])+85);
// // 	return 0;
//     fptype ret = 0;
//     fptype intg= 0;
//     fptype coefficient = 0;
//     for(int i = 1; i <= numBoxes;
//         ++i) { // Notice skip of inmost 'box' in the pyramid, corresponding to all powers zero, already accounted for.
//         fptype currTerm  = 1;
//         int currIndex    = i;
//         int sumOfIndices = 0;
// 	fptype currIntg  = 1;
// 
//         // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
//         // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
//         // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
//         // printf("[%i, %i] Start box %i %f %f:\n", BLOCKIDX, THREADIDX, i, ret, evt[8]);
//         for(int j = 0; j < numObservables; ++j) {
// //           if(sumOfIndices >= maxDegree) continue;
//             fptype x    = (evt[(indices[2 + (indices[0]) + j])]); // x, y, z...
//  	    fptype xmin = (p[(indices[4 + j])]);
//  	    fptype xmax = (p[(indices[4 + j]+1)]);
//             x=(x-xmin)/(xmax-xmin);
// //            fptype offset = (p[(indices[2 + j])]);                          // x0, y0, z0...
// //            x -= offset;
//             int currPower = currIndex % maxDegree;
//             currIndex /= maxDegree;
// //            currTerm *= pow(x, currPower);
//             bernknval = device_bernsteinkn(x,maxDegree,currPower);
//             currTerm *= bernknval.bernFunc;
// 	    currIntg *= bernknval.bernIntg;
// //            currTerm *= pow(x, currPower);
//             sumOfIndices += currPower;
//             // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
//             // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
//             // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
//             // printf("  [%f -> %f^%i = %f] (%i %i) \n", evt[indices[2 + indices[0] + j]], x, currPower, pow(x,
//             // currPower), sumOfIndices, indices[2 + indices[0] + j]);
//         }
// 
//         // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
//         // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
//         // printf(") End box %i\n", i);
//         // All threads should hit this at the same time and with the same result. No branching.
//         if(sumOfIndices >= maxDegree) continue;
// 
//         coefficient = (p[(indices[coeffNumber++])]); // Coefficient from MINUIT
//         // if ((gpuDebug & 1) && (THREADIDX == 50) && (BLOCKIDX == 3))
//         // if ((BLOCKIDX == internalDebug1) && (THREADIDX == internalDebug2))
//         // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
//         // printf("Box %i contributes %f * %f = %f -> %f\n", i, currTerm, p[indices[coeffNumber - 1]],
//         // coefficient*currTerm, (ret + coefficient*currTerm));
// 	 if ((0 == THREADIDX) && (0 == BLOCKIDX)){
// 	  printf("MultiFastBernstein MaxDegree=%d coefficient = %f   number = %d\n",maxDegree,(p[(indices[coeffNumber])]),coeffNumber-2-2*numObservables);
//       } 
//         currTerm *= coefficient;
//         currIntg *= coefficient;
//         ret += currTerm;
// 	intg+= currIntg;
//     }
//     // if ((1 > (int) floor(0.5 + evt[8])) && (gpuDebug & 1) && (paramIndices + debugParamIndex == indices))
//     // printf("Final FastBernstein: %f\n", ret);
// 
// // if (0 > ret/(intg)) ret = 0;
// // if (0 > ret) ret = -ret;
// // PADUL!!!
// // if (ret/(intg)>1) printf("Error in FastBernsteinPdf => %f",ret/(intg));
// // if (ret/(intg)<0) printf("Error in FastBernsteinPdf => %f",ret/(intg));
// //    return ret/(intg);
// //    return ret*ret/(intg*intg);
//     return ret/intg;
// //    return ret;
// }

__device__ device_function_ptr ptr_to_FastBernstein       = device_FastBernstein;
//__device__ device_function_ptr ptr_to_OffsetFastBernstein = device_OffsetFastBernstein;
__device__ device_function_ptr ptr_to_MultiFastBernstein  = device_MultiFastBernstein;

// Constructor for single-variate FastBernstein, with optional zero point.
// __host__ FastBernsteinPdf::FastBernsteinPdf(std::string n, Observable _x, std::vector<Variable> weights, unsigned int lowestDegree)
//     : GooPdf(n, _x) {
//     std::vector<unsigned int> pindices;
//     pindices.push_back(lowestDegree);
// 
//     for(auto &weight : weights) {
//         pindices.push_back(registerParameter(weight));
//     }
// 
//     GET_FUNCTION_ADDR(ptr_to_FastBernstein);
// 
//     initialize(pindices);
// }

//Constructor for single-variate FastBernstein, with optional zero point.
__host__ FastBernsteinPdf::FastBernsteinPdf(std::string n, Observable _x, std::vector<Variable> weights,std::vector<Variable> limits, unsigned int maxDegree)
    : GooPdf(n, _x) {
    std::vector<unsigned int> pindices;
    pindices.push_back(maxDegree);

    for(auto &weight : weights) {
        pindices.push_back(registerParameter(weight));
    }
    for(auto &limit : limits) {
        pindices.push_back(registerParameter(limit));
    }

     GET_FUNCTION_ADDR(ptr_to_FastBernstein);
//    GET_FUNCTION_ADDR(ptr_to_OffsetFastBernstein);

    initialize(pindices);
}
// 
 // Constructor for multivariate FastBernstein.
 __host__ FastBernsteinPdf::FastBernsteinPdf(std::string n,
				       std::vector<Observable> obses,
				       std::vector<Variable> coeffs,
				       std::vector<Variable> limits,
				       unsigned int maxDegree1,
				       unsigned int maxDegree2,
				       unsigned int maxDegree3 )
        : GooPdf(n) {
     unsigned int numParameters = 1;
 
     // For 1 observable, equal to n = maxDegree + 1.
     // For two, n*(n+1)/2, ie triangular number. This generalises:
     // 3: Pyramidal number n*(n+1)*(n+2)/(3*2)
     // 4: Hyperpyramidal number n*(n+1)*(n+2)*(n+3)/(4*3*2)
     // ...
     for(unsigned int i = 0; i < obses.size(); ++i) {
	 registerObservable(obses[i]);
//	 numParameters *= (maxDegree + 1 + i);
     }
//  
//      for(int i = observables.size(); i > 1; --i)
// 	 numParameters /= i;
 
//     int j=1;
//     numParameters = pow((maxDegree+1),coeffs.size());
     numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3+1);
     while(numParameters > coeffs.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_coeff_%i", getName().c_str(), static_cast<int>(coeffs.size()));
 
	 coeffs.emplace_back(varName, 10.,0.00001,0.,500.);
 
	 std::cout << "Warning: " << getName() << " created dummy variable " << varName
		   << "  to account for all terms.\n";
     }
 
     while(limits.size() < 2*obses.size()) {
	 char varName[100];
	 sprintf(varName, "%s_extra_limits_%i", getName().c_str(), static_cast<int>(limits.size()));
	 limits.emplace_back(varName, 0);
     }
 
     std::vector<unsigned int> pindices;
     pindices.push_back(maxDegree1);
     pindices.push_back(maxDegree2);
     pindices.push_back(maxDegree3);
 
     for(auto &limit : limits) {
	 pindices.push_back(registerParameter(limit));
     }
 
     for(auto &coeff : coeffs) {
	 pindices.push_back(registerParameter(coeff));
     }
 
     GET_FUNCTION_ADDR(ptr_to_MultiFastBernstein);
     initialize(pindices);
 }
//
 __host__ fptype FastBernsteinPdf::integrate(fptype lo, fptype hi) const {
       return 1.0;
 }

} // namespace GooFit
