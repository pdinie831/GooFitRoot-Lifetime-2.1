#include "hip/hip_runtime.h"
#include "SigmoidGausPdf.h"

EXEC_TARGET fptype device_SigmoidGaus (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2 + indices[0]]]; 
  fptype p0 = p[indices[1]];
  fptype p1 = p[indices[2]];
  fptype p2 = p[indices[3]];
  fptype p3 = p[indices[4]];
  fptype p4    = p[indices[5]];
  fptype mean  = p[indices[6]];
  fptype sigma = p[indices[7]];

  fptype ret = p0/(1+p3+ p1*exp(-p2*x))+p4*EXP(-0.5*(x-mean)*(x-mean)/(sigma*sigma));

//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("SigmoidGaus x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,alpha , ret);
//  } 

  if (ret<=0) {
       printf("SigmoidGaus <=0!!!: x = %f , p0 = %f p1 = %f f\n",x,p0,p1,p2,p3,p4,mean,sigma);
       return 0.;
  }
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_SigmoidGaus = device_SigmoidGaus; 

__host__ SigmoidGausPdf::SigmoidGausPdf (std::string n, Observable* _x, Variable* p0, Variable* p1, Variable* p2, Variable* p3, Variable* p4,
Variable* mean, Variable* sigma) 
  : GooPdf(_x, n)
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(p0));
  pindices.push_back(registerParameter(p1));
  pindices.push_back(registerParameter(p2));
  pindices.push_back(registerParameter(p3));
  pindices.push_back(registerParameter(p4));
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(sigma));
  GET_FUNCTION_ADDR(ptr_to_SigmoidGaus);
  initialise(pindices); 
}


