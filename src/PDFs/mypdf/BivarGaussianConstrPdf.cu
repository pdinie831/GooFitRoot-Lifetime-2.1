#include "hip/hip_runtime.h"
#include "BivarGaussianConstrPdf.h"

EXEC_TARGET fptype device_BivarGaussianConstr (fptype* evt, fptype* p, unsigned int* indices) {
//   fptype x = evt[indices[2 + indices[0]]]; 
//   fptype y = evt[indices[3 + indices[0]]]; 
  fptype x      = p[indices[1]];
  fptype y      = p[indices[2]];
  fptype mean1  = p[indices[3]];
  fptype sigma1 = p[indices[4]];
  fptype mean2  = p[indices[5]];
  fptype sigma2 = p[indices[6]];
  fptype rho    = p[indices[7]];
  
//  const fptype pi = 3.14159265359;
  fptype rhod = (1-rho*rho);
  if (rhod<=0) {
     printf("Error: rhod<0.!!! ==> x=%f  y=%f mean1=%f sigma1=%f mean2=%f sigma2=%f rho=%f\n",x,y,mean1,sigma1,mean2,sigma2,rho);
     return 0;
   } 
  fptype ret = (x-mean1)*(x-mean1)/(sigma1*sigma1)+(y-mean2)*(y-mean2)/(sigma2*sigma2)-2*rho*(x-mean1)*(y-mean2)/(sigma1*sigma2);
         ret = -0.5*ret/rhod;
//         ret = EXP(ret)/(2*pi*sqrt(rhod));

  //if ((0 == THREADIDX) && (0 == BLOCKIDX)) cuPrintf("BivarGaussianConstr Values %f %i %i %f %f %i\n", x, indices[1], indices[2], mean, sigma, callnumber); 
  //cuPrintf("device_BivarGaussianConstr %f %i %i %f %f %i %p %f\n", x, indices[1], indices[2], mean, sigma, callnumber, indices, ret); 
  //if ((0 == THREADIDX) && (0 == BLOCKIDX))
  //printf("device_BivarGaussianConstr %f %f %f %i %f\n", x, mean, sigma, callnumber, ret);     

//   if (ret<0) {
//      printf("Error: BivarGaussian<=0!!! ==> x=%f  y=%f mean1=%f sigma1=%f mean2=%f sigma2=%f rho=%f ret=%f\n",x,y,mean1,sigma1,mean2,sigma2,rho,ret);
//     ret=0;
//    } 
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_BivarGaussianConstr = device_BivarGaussianConstr; 

__host__ BivarGaussianConstrPdf::BivarGaussianConstrPdf (std::string n, Variable* _xdummy, Observable* _x, Observable* _y, Variable* mean1, Variable* sigma1, Variable* mean2, Variable* sigma2, Variable* rho) 
  : GooPdf(_xdummy, n) 
{
//  registerObservable(_x);
//  registerObservable(_y);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(_x));
  pindices.push_back(registerParameter(_y));
  pindices.push_back(registerParameter(mean1));
  pindices.push_back(registerParameter(sigma1));
  pindices.push_back(registerParameter(mean2));
  pindices.push_back(registerParameter(sigma2));
  pindices.push_back(registerParameter(rho));
  GET_FUNCTION_ADDR(ptr_to_BivarGaussianConstr);
  initialise(pindices); 
}

 __host__ fptype BivarGaussianConstrPdf::integrate (fptype lo, fptype hi) const {
//   //static const fptype root2 = sqrt(2.);
//   static const fptype rootPi = sqrt(atan2(0.0,-1.0));
//   static const fptype rootPiBy2 = rootPi / root2;
//   
//   unsigned int* indices = host_indices+parameters; 
//   fptype xscale = root2*host_params[indices[2]];
// 
//   /*
//   std::cout << "BivarGaussianConstr integral: " 
// 	    << xscale << " "
// 	    << host_params[indices[1]] << " "
// 	    << host_params[indices[2]] << " "
// 	    << ERF((hi-host_params[indices[1]])/xscale) << " "
// 	    << ERF((lo-host_params[indices[1]])/xscale) << " "
// 	    << rootPiBy2*host_params[indices[2]]*(ERF((hi-host_params[indices[1]])/xscale) -
// 						  ERF((lo-host_params[indices[1]])/xscale)) 
// 	    << std::endl; 
//   */
//   return rootPiBy2*host_params[indices[2]]*(ERF((hi-host_params[indices[1]])/xscale) - 
//   					    ERF((lo-host_params[indices[1]])/xscale));

  // Integral over all R. 
//   fptype sigma = host_params[indices[2]];
//   sigma *= root2*rootPi;
//   return sigma; 
return 1.;
}

