#include "hip/hip_runtime.h"
#include <goofit/PDFs/mypdf/RGaussianPdf.h>

namespace GooFit {

__device__ fptype device_RGaussian (fptype *evt, fptype *p, unsigned int *indices) {
     fptype x = evt[indices[2 + indices[0]]];
     fptype mean = p[indices[1]];
     fptype sigma = p[indices[2]];
//  fptype x     = evt[RO_CACHE(indices[2 + RO_CACHE(indices[0])])];
//  fptype mean  = RO_CACHE(p[RO_CACHE(indices[1])]);
//  fptype sigma = RO_CACHE(p[RO_CACHE(indices[2])]);

  fptype ret = exp(-0.5*(x-mean)*(x-mean)/(sigma*sigma));

  //if ((0 == THREADIDX) && (0 == BLOCKIDX)) cuPrintf("RGaussian Values %f %i %i %f %f %i\n", x, indices[1], indices[2], mean, sigma, callnumber); 
  //cuPrintf("device_RGaussian %f %i %i %f %f %i %p %f\n", x, indices[1], indices[2], mean, sigma, callnumber, indices, ret); 
  //if ((0 == THREADIDX) && (0 == BLOCKIDX))
  //printf("device_RGaussian %f %f %f %i %f\n", x, mean, sigma, callnumber, ret);     


  return ret; 
}

__device__ device_function_ptr ptr_to_RGaussian = device_RGaussian; 

__host__ RGaussianPdf::RGaussianPdf (std::string n, Observable _x, Variable mean, Variable sigma) 
  : GooPdf(n, _x) 
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(sigma));
  GET_FUNCTION_ADDR(ptr_to_RGaussian);
  initialize(pindices); 
}

__host__ fptype RGaussianPdf::integrate (fptype lo, fptype hi) const {
  //static const fptype root2 = sqrt(2.);
  static const fptype rootPi = sqrt(atan2(0.0,-1.0));
  static const fptype rootPiBy2 = rootPi / root2;
  
  unsigned int* indices = host_indices+parameters; 
  fptype xscale = root2*host_params[indices[2]];

  /*
  std::cout << "RGaussian integral: " 
	    << xscale << " "
	    << host_params[indices[1]] << " "
	    << host_params[indices[2]] << " "
	    << ERF((hi-host_params[indices[1]])/xscale) << " "
	    << ERF((lo-host_params[indices[1]])/xscale) << " "
	    << rootPiBy2*host_params[indices[2]]*(ERF((hi-host_params[indices[1]])/xscale) -
						  ERF((lo-host_params[indices[1]])/xscale)) 
	    << std::endl; 
  */
  return rootPiBy2*host_params[indices[2]]*(erf((hi-host_params[indices[1]])/xscale) - 
  					    erf((lo-host_params[indices[1]])/xscale));

  // Integral over all R. 
//   fptype sigma = host_params[indices[2]];
//   sigma *= root2*rootPi;
//   return sigma; 
}
} // namespace GooFit

