#include "hip/hip_runtime.h"
#include "SigmoidBpPdf.h"

EXEC_TARGET fptype device_SigmoidBp (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2 + indices[0]]]; 
  fptype p0 = p[indices[1]];
  fptype p1 = p[indices[2]];
  fptype p2 = p[indices[3]];
  fptype p3 = p[indices[4]];
  fptype p4 = p[indices[5]];
  fptype p5 = p[indices[6]];
  fptype p6 = p[indices[7]];
  fptype p7 = p[indices[8]];

  if (x<=0){
       printf("SigmoidBp not defined for x<=0!!!: x = %f",x);
       return 0.;
  }
  fptype ret = p2+p3*x+ p4*x*x + p5*x*x*x + p6*x*x*x*x + p7*x*x*x*x*x + p0/(1+exp(-p1*x));
//  fptype ret = p0/(1+p3+ p1*exp(-p2*x));

//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("SigmoidBp x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,alpha , ret);
//  } 

  if (ret<=0) {
       printf("SigmoidBp <=0!!!: ret = %f x = %f , p0 = %f p1 = %f p2 = %f p3 = %f p4 = %f p5 = %f p6 = %f p7 = %f\n",ret,x,p0,p1,p2,p3,p4,p5,p6,p7);
       return 0.;
  }
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_SigmoidBp = device_SigmoidBp; 

__host__ SigmoidBpPdf::SigmoidBpPdf (std::string n, Observable* _x, Variable* p0, Variable* p1, Variable* p2, Variable* p3, Variable* p4, Variable* p5, Variable* p6, Variable* p7) 
  : GooPdf(_x, n)
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(p0));
  pindices.push_back(registerParameter(p1));
  pindices.push_back(registerParameter(p2));
  pindices.push_back(registerParameter(p3));
  pindices.push_back(registerParameter(p4));
  pindices.push_back(registerParameter(p5));
  pindices.push_back(registerParameter(p6));
  pindices.push_back(registerParameter(p7));
  GET_FUNCTION_ADDR(ptr_to_SigmoidBp);
  initialise(pindices); 
}


