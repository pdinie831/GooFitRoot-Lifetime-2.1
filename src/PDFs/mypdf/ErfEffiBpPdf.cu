#include "hip/hip_runtime.h"
#include <goofit/PDFs/mypdf/ErfEffiBpPdf.h>

namespace GooFit {

__device__ fptype device_ErfEffiBp (fptype *evt, fptype *p, unsigned int *indices) {
  fptype x     = evt[indices[2 + indices[0]]]; 
  fptype p0 = p[indices[1]];
  fptype p1 = p[indices[2]];
  fptype p2 = p[indices[3]];
  fptype p3 = p[indices[4]];
  fptype p4 = p[indices[5]];
  fptype p5 = p[indices[6]];
  fptype p6 = p[indices[7]];
  fptype p7 = p[indices[8]];

  if (x<=0){
       printf("ErfEffiBp not defined for x<=0!!!: x = %f",x);
       return 0.;
  }
  fptype ret = (p2+p3*x+ p4*x*x + p5*x*x*x + p6*x*x*x*x + p7*x*x*x*x*x)*erf(p0*(x-p1));


  if (ret<=0) {
       printf("ErfEffiBp <=0!!!: ret = %f x = %f , p0 = %f p1 = %f p2 = %f p3 = %f p4 = %f p5 = %f p6 = %f p7 = %f\n",ret,x,p0,p1,p2,p3,p4,p5,p6,p7);
       return 0.;
  }
  return ret; 
}

__device__ device_function_ptr ptr_to_ErfEffiBp = device_ErfEffiBp; 

__host__ ErfEffiBpPdf::ErfEffiBpPdf (std::string n, Observable _x, Variable p0, Variable p1, Variable p2, Variable p3, Variable p4, Variable p5, Variable p6, Variable p7) 
  : GooPdf(n,_x )
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(p0));
  pindices.push_back(registerParameter(p1));
  pindices.push_back(registerParameter(p2));
  pindices.push_back(registerParameter(p3));
  pindices.push_back(registerParameter(p4));
  pindices.push_back(registerParameter(p5));
  pindices.push_back(registerParameter(p6));
  pindices.push_back(registerParameter(p7));
  GET_FUNCTION_ADDR(ptr_to_ErfEffiBp);
  initialize(pindices); 
}
} // namespace GooFit


