#include "hip/hip_runtime.h"
#include "ExpGausWithIntPdf.h"

EXEC_TARGET fptype device_ExpGausWithInt (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2 + indices[0]]]; 
  fptype mean  = p[indices[1]];
  fptype sigma = p[indices[2]];
  fptype tau = p[indices[3]];

  fptype ret = 0.5*tau; 
  fptype exparg = ret * (2*mean + tau*sigma*sigma - 2*x);
  fptype erfarg = (mean + tau*sigma*sigma - x) / (sigma * 1.4142135623);

  ret *= EXP(exparg); 
  ret *= ERFC(erfarg); 
//  if ((0 == THREADIDX) && (0 == BLOCKIDX))
//  printf("ExpGausWithIntPdf x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,alpha , ret);
//  return 0.;

  if (ret<=0) {
       printf("ExpGausWithIntPdf <=0!!!: x = %f , mean = %f sigma = %f alpha = %f\n",x,mean,sigma,tau);
//       return 0.;
  }
  fptype lo = 0.0003;
  fptype hi = 0.007;
  fptype intg_hi = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - hi))*
                   erfc((tau+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(tau*(tau*sigma*sigma/2.+mean - lo))*
                   erfc((tau+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));   
//   fptype u_hi = tau * (hi - mean);
//   fptype v_hi = tau * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// 
//   fptype u_lo = tau * (lo - mean);
//   fptype v_lo = tau * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));

  fptype integral = (intg_hi-intg_lo);
  return ret/integral; 
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausWithInt = device_ExpGausWithInt; 

ExpGausWithIntPdf::ExpGausWithIntPdf (std::string n, Observable* _x, Variable* mean, Variable* sigma, Variable* tau) 
  : GooPdf(_x, n)
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(sigma));
  pindices.push_back(registerParameter(tau));
  GET_FUNCTION_ADDR(ptr_to_ExpGausWithInt);
  initialise(pindices); 
}

 __host__ fptype ExpGausWithIntPdf::integrate (fptype lo, fptype hi) const {
/*  unsigned int* indices = host_indices+parameters; 
 fptype mean = host_params[indices[1]]  ;
 fptype sigma= host_params[indices[2]]; 
 fptype tau  = host_params[indices[3]]  ;
 fptype u_hi = tau * (hi - mean);
 fptype v_hi = tau *sigma;
// fptype v_hi = tau * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = tau * (lo - mean);
 fptype v_lo = tau * sigma;
// fptype v_lo = tau * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 return (intg_hi-intg_lo);
 */
   return 1.;
 }

