#include "hip/hip_runtime.h"
//
//
//
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/stat.h>
#include "Riostream.h"
#include <map>
#include <string>
//#include <boost/algorithm/string.hpp>
//#include <boost/algorithm/string/trim.hpp>
#include <vector>
#include <math.h>
//#include <TCint.h>
#include <TGenericClassInfo.h> 
#include <TH1.h>
#include <TH2.h>
#include <TH1F.h>
#include <TH2F.h>
#include <TH1D.h>
#include <TH2D.h>
#include <TROOT.h>
#include <TEnv.h>
#include <TSystem.h>
#include <TTree.h>
#include "TBranch.h"
#include <TApplication.h>
#include <TFile.h>
#include <TCanvas.h>
#include <TStyle.h> 
#include <TF1.h>  
#include <TF2.h> 
#include <TLorentzVector.h>
#include <TVector3.h>
#include "TDSet.h"
#include "TChain.h"
#include <time.h> 
#include <TSystemDirectory.h>
#include <TLegend.h>
#include <TMinuit.h>

#include "TRandom.h" 
#include  <TStopwatch.h>
#include "TH1F.h"
#include "TH2F.h"			// unused?
#include "TStyle.h"
#include "TCanvas.h"

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>
// GooFit stuff
#include <goofit/Application.h>
#include <goofit/FitManager.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/PDFs/basic/GaussianPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>
#include <goofit/PDFs/combine/ConvolutionPdf.h>
#include <goofit/PDFs/combine/EventWeightedAddPdf.h>
#include <goofit/PDFs/combine/MappedPdf.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/fitting/FitManagerMinuit1.h>
#include <goofit/Variable.h>
#include <goofit/PDFs/mypdf/RGaussianPdf.h>
#include <goofit/PDFs/mypdf/ErfcMassPdf.h>
#include <goofit/PDFs/mypdf/SimpleCheby2Pdf.h> 
#include <goofit/PDFs/mypdf/ExpGausProdBPdf.h>
#include <goofit/PDFs/mypdf/ErfcPolyPdf.h>
#include <goofit/PDFs/mypdf/ErfEffiBpPdf.h>
//#include "ExpGausPEEPdf.h" 
//#include "ExpGausMPdf.h" 
//#include "ExpGausWithIntPdf.h"
//#include "ExpGausPEEfixSigmaPdf.h" 
//#include "ExpGausProdBPdf.h"
//#include "ExpGausProdEffiBPdf.h"
//#include "ExpGausPEESigmaBPdf.h" 
//#include "PolyEffiPdf.h" 
//#include "ErfcPolyPdf.h"
//#include "ErfcMassPdf.h"
//#include "SigmoidBpPdf.h"
//#include <goofit/PDFs/mypdf/SigmoidBpPdf.h>
//#include "ErfEffiBpPdf.h"
//#include "SigmoidGausPdf.h"
//#include "GooFit/BivarGaussianConstrPdf.h"
//#include "GooFit/TrivarGaussianConstrPdf.h"
// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 

using namespace std; 
using namespace GooFit;
using namespace ROOT;
void fitTauSBModel();
GooFit::Application *app_ptr;
bool minuit1;


int main (int argc, char** argv) {

  TApplication tapp("TApp",&argc, argv);
  GooFit::Application app("testGoofit3DBp-2016 fit example", argc, argv);
  app_ptr = &app;
//  app.require_subcommand();

//  app.add_flag("--minuit1", minuit1, "Use Minuit 1 instead of Minuit 2");
 
  TStopwatch TimeWatch;
  TimeWatch.Start();

  fitTauSBModel(); 
//  app.Run() ;
  cout<<"esco..." <<endl;
  TimeWatch.Stop();
  TimeWatch.Print();
  
  GOOFIT_PARSE(app);
  return 0 ;
}


void fitTauSBModel(){


  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);

   gROOT->Reset();
   gROOT->SetStyle("Plain");
   gROOT->ForceStyle();
   gStyle->SetOptStat(000000);
   gStyle->SetOptFit(000000);


   TCanvas* c1 = new TCanvas("c1","Mass PLOTS",200,10,900,780);
   TCanvas* c2 = new TCanvas("c2","cTau PLOTS",200,10,900,780);
   TCanvas* c3 = new TCanvas("c3","STau PLOTS",200,10,900,780);
//   TCanvas*c2 = new TCanvas("c2","PLOTS",200,10,900,780);
   //TPad* pad1 = (TPad*)c1->GetPad(0);
//   TPad* pad2 = (TPad*)c2->GetPad(0);
   //pad1->SetLeftMargin(0.15); 
//   pad2->SetLeftMargin(0.15); 




//  Char_t    InputFileName[300] = "rooFit/Ntuple2012BPJpsiK_RD-SaraCuts-2016-MC.root";
//    Char_t    InputFileName[300] = "rooFit/test-Run2016G-Charmonium-v1.root";
//Char_t    InputFileName[300] = "rooFit/test-Run2016H-Charmonium-v2.root";
//  Char_t    InputFileName[300] = "testproof2DCut-Bp-SaraCuts-NewCutPDGMass.root";
//  Char_t    InputFileName[300] = "testproof2DCut-Run2016-SaraCuts-NewCutPDGMass-TriggerMatchMuons.root";
//      Char_t    InputFileName[300] = "testproof2DCut-Run2016-SaraCuts-NewCutPDGMass-Els3-testMu4.root";
      Char_t    InputFileName[300] = "testproof2DCut-Run2016-SaraCuts-NewCutPDGMass-Els3-testMu4-TrigMatch.root";
//      Char_t    InputFileName[300] = "testproof2DCut-Run2016-SaraCuts-NewCutPDGMass.root";
//    Char_t    InputFileName[300] = "testproof2DCut-RunHCDF2016-SaraCuts-NewCutPDGMass.root
  Char_t    InputTauBpTreeName[10]   = "TauBpTree";
  TFile*InputFile = TFile::Open(InputFileName,"READ","ROOT file");
  
   Char_t    OutFileName[300] =  "testGoofit3DBp-2016.root";
   gSystem->Exec(Form("mv %s %s.tmp",OutFileName,OutFileName));
   TFile*OutFile = TFile::Open(OutFileName,"RECREATE");
   
   int PlotLineWidth = 1;
//   float PlotLineWidth = 1.2;
   float MarkerSize    = 0.35;
 
  double xBpMass;
//  double xBpTau;
  double xBpcTau;
  double xSBpcTau;
//  double c_const       = 0.0299792458;
//     double XMinSign = 5.12;
//     double XMaxSign = 5.44;
  double XMinSign = 5.1;
  double XMaxSign = 5.45;
  double BpMass   = 5.2784;
//double BpSigma  = 0.020;
//  double BpSigma  = 0.022;
  double BpSigma  = 0.028;

//  double NSigmaSB = 3.6;
  double NSigmaSBL = -2.;
  double NSigmaSBR = 0;
  double BiasSB   = 6;
 
//      double XMinSign = 5.12;
//      double XMaxSign = 5.44;
//double XMinSign = 5.15;
//double XMaxSign = 5.40;
//   double XMinSBL = 4.879000;
//   double XMaxSBL = 5.159000;
//   double XMinSBR = 5.399000;
//   double XMaxSBR = 5.679000;
  double XMinSBL = XMinSign - (BiasSB+NSigmaSBL)*BpSigma;
  double XMaxSBL = BpMass   -  BiasSB           *BpSigma;
//  double XMaxSBL = BpMass - BiasSB          *BpSigma;
  double XMinSBR = BpMass   +  BiasSB           *BpSigma;
  double XMaxSBR = XMaxSign + (BiasSB+NSigmaSBR)*BpSigma;
//  double XMaxSBR = BpMass +(BiasSB+NSigmaSB)*BpSigma;
  double XMin = 0.004;
  double XMax = 0.30;
  double SXMin = 0.0003;
  double SXMax = 0.005;
//  double SXMax = 0.0079;
//  double SXMax = 0.0079;
  double XStepSign = 0.001;
  double XStepcTau = 0.001;
//  double XStepScTau = 0.00002;
  double XStepMinuit = 0.00001;
  double XHScale = 4;
  
  double c_const       = 0.0299792458;

  char strbuffer[1000];


  printf("(xBpMass>%8f && xBpMass<%8f || xBpMass>%8f && xBpMass<%8f)\n",XMinSBL,XMaxSBL,XMinSBR,XMaxSBR);

//  GooFit::Observable* xMass  = new GooFit::Observable("xMass",XMinSign, XMaxSign); 
  GooFit::Observable xMass("xMass",XMinSign, XMaxSign); 
  xMass.setNumBins( (XMaxSign -XMinSign)/XStepSign );
  TH1F HxMass( "HxMass" , "B^{+} Mass"    ,	     xMass.getNumBins(), xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1F pdfHist("pdfHist", "B^{+} Mass Fit",  XHScale*xMass.getNumBins(), xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1F sigHist("sigHist", "B^{+} Mass Fit",  XHScale*xMass.getNumBins(), xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1F bkgHist("bgkHist", "B^{+} Mass Fit",  XHScale*xMass.getNumBins(), xMass.getLowerLimit(), xMass.getUpperLimit());

//  GooFit::Observable* xcTau  = new GooFit::Observable("xcTau",XMin, XMax); 
  GooFit::Observable xcTau("xcTau",XMin, XMax); 
  int NBINS = (XMax -XMin)/XStepcTau;
  xcTau.setNumBins(NBINS);
  GooFit::Observable xScTau("xScTau",SXMin, SXMax); 
  xScTau.setNumBins(NBINS);
  std::cout<<"xMass.getNumBins() = "<<xMass.getNumBins()<<std::endl;
  std::cout<<"xcTau.getNumBins() = "<<xcTau.getNumBins()<<std::endl;
  std::cout<<"xScTau.getNumBins() = "<<xScTau.getNumBins()<<std::endl;
  std::cout<<"xScTau.getNumBins() = "<<xScTau.getNumBins()<<std::endl;
 
  TH1F HxcTau(    "HxcTau"   , "B^{+} cTau",          xcTau.getNumBins(), xcTau.getLowerLimit(),  xcTau.getUpperLimit());
  TH1F HxcTauSB(  "HxcTauSB" , "B^{+} cTau SB",       xcTau.getNumBins(), xcTau.getLowerLimit(),  xcTau.getUpperLimit());
  TH1F HxScTau(   "HxScTau"  , "B^{+} cTau Sigma",    xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
  TH1F HxScTauSB( "HxScTauSB", "B^{+} cTau Sigma SB", xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
//   TH1F pdf_cTau_Hist( "pdf_cTau_Hist" , "B+ cTau model   pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit());
//   TH1F sig_cTau_Hist( "sig_cTau_Hist" , "B+ cTau signal  pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit());
//   TH1F bkg_cTau_Hist( "bkg_cTau_Hist" , "B+ cTau bckg    pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit());
//   TH1F pdf_STau_Hist( "pdf_STau_Hist" , "B+ ScTau model  pdf",    xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
//   TH1F sig_STau_Hist( "sig_STau_Hist" , "B+ ScTau signal pdf",    xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
//   TH1F bkg_STau_Hist( "bkg_STau_Hist" , "B+ ScTau bckg   pdf",    xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
// 
//   TH1F pdf_cTau_Hist2D( "pdf_cTau_Hist2D" , "B+ cTau model   pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit());
//   TH1F sig_cTau_Hist2D( "sig_cTau_Hist2D" , "B+ cTau signal  pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit());
//   TH1F bkg_cTau_Hist2D( "bkg_cTau_Hist2D" , "B+ cTau bckg    pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit());              

//  TH2F pdf_cTauSTau_Hist2D( "pdf_cTauSTau_Hist2D" , "B+ cTau model   pdf",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit(), xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
//  TH2F sig_cTauSTau_Hist2D( "sig_cTauSTau_Hist2D" , "B+ cTau model   sig",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit(), xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
//  TH2F bkg_cTauSTau_Hist2D( "bkg_cTauSTau_Hist2D" , "B+ cTau model   bkg",    xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit(), xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
  TH2F pdf_cTauSTau_Hist2D( "pdf_cTauSTau_Hist2D" , "Bc+ cTau model   pdf",  XHScale*xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit(), XHScale*xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
  TH2F sig_cTauSTau_Hist2D( "sig_cTauSTau_Hist2D" , "Bc+ cTau model   sig",  XHScale*xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit(), XHScale*xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
  TH2F bkg_cTauSTau_Hist2D( "bkg_cTauSTau_Hist2D" , "Bc+ cTau model   bkg",  XHScale*xcTau.getNumBins(), xcTau.getLowerLimit(), xcTau.getUpperLimit(), XHScale*xScTau.getNumBins(), xScTau.getLowerLimit(), xScTau.getUpperLimit());
//
// Mass Spectrum
//
  GooFit::Variable mean  ("mean"  ,5.2785,XStepMinuit, 5., 5.5);
  GooFit::Variable sigma1("sigma1",0.0139,XStepMinuit, 0., 1.);
  GooFit::Variable sigma2("sigma2",0.0228,XStepMinuit, 0., 1.);
  GooFit::Variable sigma3("sigma3",0.0601,XStepMinuit, 0., 1.);

//   GaussianPdf* gauss1 = new GaussianPdf("gauss1", xMass, mean, sigma1);
//   GaussianPdf* gauss2 = new GaussianPdf("gauss2", xMass, mean, sigma2);
//   GaussianPdf* gauss3 = new GaussianPdf("gauss3", xMass, mean, sigma3);
  RGaussianPdf* gauss1 = new RGaussianPdf("gauss1", xMass, mean, sigma1);
  RGaussianPdf* gauss2 = new RGaussianPdf("gauss2", xMass, mean, sigma2);
  RGaussianPdf* gauss3 = new RGaussianPdf("gauss3", xMass, mean, sigma3);

/*   GooFit::Variable* meanBckgBp   = new GooFit::Variable("meanBckgBp" ,5.360,0.00001, 5., 5.5);
  GooFit::Variable* sigmaBckgBp  = new GooFit::Variable("sigmaBckgBp",0.030,0.00001, 0. , 1. );
  GooFit::Variable* meanBckgB0   = new GooFit::Variable("meanBckgB0" ,5.090,0.00001, 5. , 5.2);
  GooFit::Variable* sigmaBckgB0  = new GooFit::Variable("sigmaBckgB0",0.025,0.00001, 0. , 1.);
 */  
//   GooFit::Variable* meanBckgBp   = new GooFit::Variable("meanBckgBp" ,5.37 ,0,  5.2, 5.7);
//   GooFit::Variable* sigmaBckgBp  = new GooFit::Variable("sigmaBckgBp",0.033,0,  0.013 , 0.05 );
//   GooFit::Variable* meanBckgB0   = new GooFit::Variable("meanBckgB0" ,5.090,0,  5.0 , 5.15);
//   GooFit::Variable* sigmaBckgB0  = new GooFit::Variable("sigmaBckgB0",0.025,0,  0.01 , 0.05);
//   GooFit::Variable meanBckgBp ("meanBckgBp" ,5.35129e+00,0., 5.25, 5.45);
//   GooFit::Variable sigmaBckgBp("sigmaBckgBp",2.10790e-02,0., 0.013 , 0.04 );
  GooFit::Variable meanBckgBp ("meanBckgBp" ,5.35129e+00);
  GooFit::Variable sigmaBckgBp("sigmaBckgBp",2.10790e-02);
//  GooFit::Variable* meanBckgB0   = new GooFit::Variable("meanBckgB0" ,5.010,0., 5.0 , 5.15);
//   GooFit::Variable meanBckgB0 ("meanBckgB0" ,5.10,0., 5.0. , 5.15);
//   GooFit::Variable sigmaBckgB0("sigmaBckgB0",0.029,0., 0.02 , 0.04);
  GooFit::Variable meanBckgB0 ("meanBckgB0" ,5.10 );
  GooFit::Variable sigmaBckgB0("sigmaBckgB0",0.029);
  GooFit::Variable	   wb1("wb1",1.75628e-01, 0., 1.);
  GooFit::Variable	   wb2("wb2",1.17246e-01, 0., 1.);
  GooFit::Variable	   wb3("wb3",0.1, 0., 1.);

  RGaussianPdf* gaussBckgBp = new RGaussianPdf("gaussBckgBp", xMass, meanBckgBp, sigmaBckgBp);
  RGaussianPdf* gaussBckgB0 = new RGaussianPdf("gaussBckgB0", xMass, meanBckgB0, sigmaBckgB0);
  

  GooFit::Variable* wg1 = new GooFit::Variable("wg1",0.44, 0., 1.);
  GooFit::Variable* wg2 = new GooFit::Variable("wg2",0.5 , 0., 1.);
  GooFit::Variable* signalYield = new GooFit::Variable("signalYield",8.88646e+05,      100000., 2000000.);
  GooFit::Variable* bckgYield   = new GooFit::Variable("bckgYield"  ,1.25481e+05 ,      10000., 400000.);

 
//   GooFit::Variable* constaCoef = new GooFit::Variable("constaCoef", 70., XStepMinuit, 20., 1000); 
//   GooFit::Variable* linearCoef = new GooFit::Variable("linearCoef", 0.1, XStepMinuit, -3.5, 10.); 
//   GooFit::Variable* secondCoef = new GooFit::Variable("secondCoef", 0.1, XStepMinuit, 0, 10);
//   GooFit::Variable* thirdCoef  = new GooFit::Variable("thirdCoef" , 0.1, XStepMinuit, 0, 10);
//  GooFit::Variable* constaCoef = new GooFit::Variable("constaCoef", 1. ,XStepMinuit,0.,1000. ); 
//  GooFit::Variable* linearCoef = new GooFit::Variable("linearCoef", 0.001,XStepMinuit,0,10 ); 

//  GooFit::Variable* p0   = new GooFit::Variable("p0", -2.31800e-01,-10.,10. ); 
//  GooFit::Variable  p1("p1",-2.78404e-01,-10.,10. ); 
  GooFit::Variable p0("p0",-3.08433e-01,-10.,10. ); 
  GooFit::Variable p1("p1"          ,0.,-10.,10. ); 
  GooFit::Variable VMinSign("VMinSign",XMinSign ); 
  GooFit::Variable VMaxSign("VMaxSign",XMaxSign ); 
  SimpleCheby2Pdf* SimpleCheby2  = new SimpleCheby2Pdf("SimpleCheby2", xMass, p0, p1,VMinSign,VMaxSign);

// double  fullRange = XMaxSign - XMinSign;
// double  minScaled = -1. + 2. * (XMinSign - xminfull) / fullRange;
// 
// double  maxScaled = +1. - 2. * (xmaxfull - XMaxSign)) / fullRange; 

//  GooFit::Variable* aslope     = new GooFit::Variable("slope", -1.);
  //GooFit::Variable* aslope     = new GooFit::Variable("slope", 0.39, -10, 10);
  //GooFit::Variable* apower     = new GooFit::Variable("apower", 6, 0, 10);
//  GooFit::Variable* apower     = new GooFit::Variable("apower", 1.18, XStepMinuit, 0.9, 15.);
//  GooFit::Variable* apower     = new GooFit::Variable("apower", 1.18, XStepMinuit, 0.9, 6.);
//  GooFit::Variable* apower     = new GooFit::Variable("apower", 1.18, 0.001, 0.9, 5.);
  //GooFit::Variable* treshold   = new GooFit::Variable("treshold" ,5.168,XStepMinuit, 5.02, 6.);
//  GooFit::Variable* treshold   = new GooFit::Variable("treshold" ,5.33,0, 5.04, 6.);

 
 
  std::vector<GooFit::Variable> weightsSignalMass;
  weightsSignalMass.push_back(*wg1);
//  weightsSignalMass.push_back(wg2); 

  std::vector<PdfBase*> compsSignalMass;
  compsSignalMass.push_back(gauss1);
  compsSignalMass.push_back(gauss2);
//  compsSignalMass.push_back(gauss3);
  string str = "signalMass";
  sprintf(strbuffer, "signalMass");
  GooFit::AddPdf signalMass(str, weightsSignalMass, compsSignalMass); 
//  GooFit::AddPdf * signalMass = new GooFit::AddPdf(str, weightsSignalMass, compsSignalMass); 
//  signalMass.addSpecialMask(PdfBase::ForceCommonNorm) ;

//  vector<GooFit::Variable*> weightsPoly;
//  weightsPoly.push_back(constaCoef);
//  weightsPoly.push_back(linearCoef);
//  weightsPoly.push_back(secondCoef);
//  weightsPoly.push_back(thirdCoef);

  
//  PolynomialPdf* polyTmp = new PolynomialPdf("polyTmp", xMass, weightsPoly); 
//   std::vector<PdfBase*> compsPoly2;
//   compsSignalMass.push_back(polyTmp);
//   compsSignalMass.push_back(polyTmp);
//   
//   ProdPdf* poly      = new ProdPdf("poly"  ,compsPoly2 );
 
//  PolynomialPdf* poly = new PolynomialPdf("poly", xMass, weightsPoly); 

//   vector<GooFit::Variable*> weightsErfcMassBckg;
//   weightsErfcMassBckg.push_back(constaCoef);
//   weightsErfcMassBckg.push_back(linearCoef);
//   weightsErfcMassBckg.push_back(secondCoef);
//   weightsErfcMassBckg.push_back(thirdCoef);
  GooFit::Variable ps0("ps0",4.44421e+01,  8.5 , 120.); 
  GooFit::Variable ps1("ps1",5.13630e+00,   5.1,5.2);
  GooFit::Variable ps2("ps2",1.); 
  GooFit::Variable ps3("ps3",0.);
  ErfcMassPdf* ErfcMassBckg = new ErfcMassPdf("ErfcMassBckg",xMass,ps0,ps1,ps2,ps3);
  
  std::vector<GooFit::Variable> weightsBckgMass;
  weightsBckgMass.push_back(wb1);
  weightsBckgMass.push_back(wb2);
//  weightsBckgMass.push_back(wb3);

//  ArgusPdf* argus = new  ArgusPdf("argus", xMass, treshold, aslope, true, apower);  

  std::vector<PdfBase*> compsBckgMass;
  compsBckgMass.push_back(gaussBckgBp);
//  compsBckgMass.push_back(gaussBckgB0);
//    compsBckgMass.push_back(argus);
//  compsBckgMass.push_back(poly);
  compsBckgMass.push_back(SimpleCheby2);
  compsBckgMass.push_back(ErfcMassBckg);
 
  GooFit::AddPdf bckgMass("bckgMass", weightsBckgMass, compsBckgMass);
//  bckgMass.addSpecialMask(PdfBase::ForceCommonNorm) ;

  
//==============================================================================
//==============================================================================
//==============================================================================
// Lifetime
//==============================================================================
//==============================================================================
//==============================================================================

     GooFit::Variable cTau ("cTau"  ,1./(1.638 *c_const), 10., 30.);
//     GooFit::Variable tauSB1("tauSB1",1./(1.620 *c_const), 0., 1000.);
//     GooFit::Variable tauSB2("tauSB2",1./(0.400 *c_const), 0., 1000.);

     GooFit::Variable tauSB1("tauSB1",1/6.31009e-03, 0., 2000.);
     GooFit::Variable tauSB2("tauSB2",1/4.72861e-02, 0., 1000.);
     GooFit::Variable tauSB3("tauSB3",1/4.71086e-02, 0., 1000.);
     

//GooFit::Variable cTau  ("cTau"  ,1./( 1.638 *c_const),0.00, 1000.);
//GooFit::Variable tauSB1("tauSB1",1./( 1.440 *c_const),0.00, 1000.);
//GooFit::Variable tauSB2("tauSB2",1./( 1.600 *c_const),0.00, 1000.);

  GooFit::Variable meanResSign ("meanResSign"  ,0.);

  GooFit::Variable meanResBckg1("meanResBckg1" ,0.);
  GooFit::Variable meanResBckg2("meanResBckg2" ,0.);
//  GooFit::Variable meanResBckg2("meanResBckg2" ,0.,0,0.01);
  GooFit::Variable meanResBckg3("meanResBckg3" ,0.);

  GooFit::Variable sigmaRes    ("sigmaRes"	,0.0003,0,001);

  GooFit::Variable sigmaResBckg("sigmaResBckg"  ,0.0003,0,001);



//  GooFit::Variable* meanLandauErrSign      = new GooFit::Variable( "meanLandauErrSign"        ,0.0015 ,SXMin, SXMax);
//  GooFit::Variable* sigmaLandauErrorSign   = new GooFit::Variable( "sigmaLandauErrorSign"     ,0.0002 ,SXMin, SXMax);

//  GooFit::Variable* meanLandauErrBckg      = new GooFit::Variable( "meanLandauErrBckg"        ,0.0015 ,SXMin, SXMax);
//  GooFit::Variable* sigmaLandauErrorBckg   = new GooFit::Variable( "sigmaLandauErrorBckg"     ,0.0002 ,SXMin, SXMax);
 
//  GooFit::Variable* meanGaussianErrSign    = new GooFit::Variable( "meanGaussianErrSign"      ,0.0013  ,SXMin, SXMax);
//  GooFit::Variable* sigmaGaussianErrorSign = new GooFit::Variable( "sigmaGaussianErrorSign"   ,0.0003  ,0.00001, SXMax);

//  GooFit::Variable* meanGaussianErrBckg1   = new GooFit::Variable( "meanGaussianErrBckg1"      ,0.0013  ,0.,SXMin, SXMax);
//  GooFit::Variable* sigmaGaussianErrorBckg1= new GooFit::Variable( "sigmaGaussianErrorBckg1"   ,0.0003  ,0.,0.00001, SXMax);

//  GooFit::Variable* meanBifurGErrSign      = new GooFit::Variable( "meanBifurGErrSign"        ,0.0015 ,0., SXMax);
//  GooFit::Variable* sigmaLBifurGErrSign    = new GooFit::Variable( "sigmaLBifurGErrSign"      ,0.0003 ,0.00001, SXMax);
//  GooFit::Variable* sigmaRBifurGErrSign    = new GooFit::Variable( "sigmaRBifurGErrSign"      ,0.0009 ,0.00001, SXMax);

//  GooFit::Variable* meanBifurGErrBckg      = new GooFit::Variable( "meanBifurGErrBckg"	      ,0.0015 ,0., SXMax);
//  GooFit::Variable* sigmaLBifurGErrBckg    = new GooFit::Variable( "sigmaLBifurGErrBckg"      ,0.0003 ,0.00001, SXMax);
//  GooFit::Variable* sigmaRBifurGErrBckg    = new GooFit::Variable( "sigmaRBifurGErrBckg"      ,0.0009 ,0.00001, SXMax);
  
//   GooFit::Variable* tauErrSign   = new GooFit::Variable("tauErrSign",2100,0.,0., 10000.);
//   GooFit::Variable* tauErrBckg1  = new GooFit::Variable("tauErrBckg1",2100,0.,0., 10000.);

//   GooFit::Variable* meanGaussianErrSign1    = new GooFit::Variable( "meanGaussianErrSign1"     ,1.20736e-03  ,0.0001, SXMax);
//   GooFit::Variable* meanGaussianErrSign2    = new GooFit::Variable( "meanGaussianErrSign2"     ,1.66720e-03  ,0.0001, SXMax);
//   GooFit::Variable* meanGaussianErrSign3    = new GooFit::Variable( "meanGaussianErrSign3"     ,1.72100e-03  ,0.0001, SXMax);
  GooFit::Variable meanGaussianErrSign1( "meanGaussianErrSign1"     ,1.19914e-03  ,0.0001, SXMax);
  GooFit::Variable meanGaussianErrSign2( "meanGaussianErrSign2"     ,1.66008e-03  ,0.0001, SXMax);
  GooFit::Variable meanGaussianErrSign3( "meanGaussianErrSign3"     ,1.72100e-03  ,0.0001, SXMax);

//   GooFit::Variable* sigmaGaussianErrorSign1 = new GooFit::Variable( "sigmaGaussianErrorSign1"  ,2.57463e-04  ,0.0001, 0.001);
//   GooFit::Variable* sigmaGaussianErrorSign2 = new GooFit::Variable( "sigmaGaussianErrorSign2"  ,2.82726e-04  ,0.0001, 0.001);
//   GooFit::Variable* sigmaGaussianErrorSign3 = new GooFit::Variable( "sigmaGaussianErrorSign3"  ,4.55213e-04  ,0.0001, 0.001);
  GooFit::Variable sigmaGaussianErrorSign1( "sigmaGaussianErrorSign1"  ,2.83235e-04  ,0.00001, 0.001);
  GooFit::Variable sigmaGaussianErrorSign2( "sigmaGaussianErrorSign2"  ,2.83263e-04  ,0.00001, 0.001);
  GooFit::Variable sigmaGaussianErrorSign3( "sigmaGaussianErrorSign3"  ,4.55213e-04  ,0.00001, 0.001);

  GooFit::Variable meanGaussianErrBckg1   ( "meanGaussianErrBckg1"	,1.51119e-03 ,0.00005, 0.005);
  GooFit::Variable meanGaussianErrBckg2   ( "meanGaussianErrBckg2"	,1.51338e-03 ,0.00005, 0.005);
  GooFit::Variable meanGaussianErrBckg3   ( "meanGaussianErrBckg3"	,1.86837e-03 ,0.00005, 0.005);

  GooFit::Variable sigmaGaussianErrorBckg1( "sigmaGaussianErrorBckg1"	,3.65925e-04,0.0001, 0.001);
  GooFit::Variable sigmaGaussianErrorBckg2( "sigmaGaussianErrorBckg2"	,3.67023e-04,0.0001, 0.001);
  GooFit::Variable sigmaGaussianErrorBckg3( "sigmaGaussianErrorBckg3"	,3.17983e-04,0.0001, 0.001);


//   GooFit::Variable* tauErrSign1  = new GooFit::Variable("tauErrSign1" ,1.49315e+03,1000., 20000.);
//   GooFit::Variable* tauErrSign2  = new GooFit::Variable("tauErrSign2" ,3.39595e+03,1000., 11000.);
  GooFit::Variable tauErrSign1("tauErrSign1" ,3.38691e+03,1000., 22000.);
  GooFit::Variable tauErrSign2("tauErrSign2" ,1.49315e+04,1000., 22000.);
  GooFit::Variable tauErrSign3("tauErrSign3" ,1.02715e+04,1000., 11000.);

  GooFit::Variable tauErrBckg1("tauErrBckg1",1/3.78093e-04, 2000., 18000.);
  GooFit::Variable tauErrBckg2("tauErrBckg2",1/3.71851e-04, 1600., 20000.);
  GooFit::Variable tauErrBckg3("tauErrBckg3",1/2.76171e-04, 2000., 10000);

  GooFit::Variable wt1("wt1",3.17124e-02, 0., 1.);
  GooFit::Variable wt2("wt2",0.05, 0., 1.);
//Break
//GenTrue break 5 sigma 0.01-0.35
//  double ef0Param = 8.86226e-02       ;
//  double ef1Param = 2.45898e+02       ;
//  double ef2Param =-5.56651e-02       ;
//  double ef3Param =-2.09280e-01       ;
//  double ef4Param = 3.23734e+00       ;
//  double ef5Param =-2.48160e+01       ;
//  double ef6Param = 7.36517e+01       ;
//  double ef7Param =-7.46837e+01       ;
//GenTrue break 5 sigma 0.006-0.35 TriggerMatchMuons
// 	 double ef0Param =   	1.66702e-01     ;
// 	 double ef1Param =   	3.02277e+02     ;
// 	 double ef2Param =     -1.34226e-01     ;
// 	 double ef3Param =     -1.89685e-01     ;
// 	 double ef4Param =   	2.93533e+00     ;
// 	 double ef5Param =     -2.27782e+01     ;
// 	 double ef6Param =   	6.74533e+01     ;
// 	 double ef7Param =     -6.77444e+01     ;
// 	     double ef0Param =  1.66612e-01  	;
// 	     double ef1Param =  3.02190e+02  	;
// 	     double ef2Param = -1.34140e-01  	;
// 	     double ef3Param = -1.89753e-01  	;
// 	     double ef4Param =  2.93743e+00  	;
// 	     double ef5Param = -2.27969e+01  	;
// 	     double ef6Param =  6.75209e+01  	;
// 	     double ef7Param = -6.78281e+01  	;
//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.015-0.35 Els>3
//
// 	   double ef0Param =   2.62140e-02     ;
// 	   double ef1Param =   8.93091e+01     ;
// 	   double ef2Param =   1.34002e-02     ;
// 	   double ef3Param =  -4.58456e-01     ;
// 	   double ef4Param =   6.60342e+00     ;
// 	   double ef5Param =  -4.54769e+01     ;
// 	   double ef6Param =   1.32014e+02     ;
// 	   double ef7Param =  -1.36165e+02     ;
// 	   double sef0Param =  2.18070e-03      ;
// 	   double sef1Param =  8.29254e+00      ;
//            double rhoParam  =  0.714	        ;
//  	   double sef2Param =  2.35149e-03      ;
//   GooFit::Variable* sef11 = new GooFit::Variable("sef11", 4.755e-06 ); 
//   GooFit::Variable* sef22 = new GooFit::Variable("sef22", 6.877e+01 ); 
//   GooFit::Variable* sef33 = new GooFit::Variable("sef33", 5.529e-06 ); 
//   GooFit::Variable* sef12 = new GooFit::Variable("sef12", 1.292e-02 ); 
//   GooFit::Variable* sef13 = new GooFit::Variable("sef13",-4.998e-06 ); 
//   GooFit::Variable* sef23 = new GooFit::Variable("sef23",-1.649e-02 ); 
//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.006-0.35 Els>3 abs(dY)<1.2
// 
// 	   double ef0Param =   1.29837e-01  	;
// 	   double ef1Param =   3.11998e+02  	;
// 	   double ef2Param =  -1.05559e-01  	;
// 	   double ef3Param =  -1.28769e-01  	;
// 	   double ef4Param =   2.00568e+00  	;
// 	   double ef5Param =  -1.55316e+01  	;
// 	   double ef6Param =   4.48051e+01  	;
// 	   double ef7Param =  -4.31350e+01  	;
//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.006-0.35 Els>3 abs(dY)<1.6

// 	   double ef0Param =   1.57339e-01  	;
// 	   double ef1Param =   3.06289e+02  	;
// 	   double ef2Param =  -1.27415e-01  	;
// 	   double ef3Param =  -1.77634e-01  	;
// 	   double ef4Param =   2.77372e+00  	;
// 	   double ef5Param =  -2.15556e+01  	;
// 	   double ef6Param =   6.40929e+01  	;
// 	   double ef7Param =  -6.47876e+01  	;
//    	   double sef0Param =	 3.30840e-03	;
//    	   double sef1Param =	 3.85188e+00	;
//      	   double rhoParam  =	 0.893  	;
// 	   double sef2Param =   3.35169e-03     ;
// 
//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.006-0.35 Els>3
// 	   double ef0Param =  1.69773e-01	  ;
// 	   double ef1Param =  3.04705e+02	  ;
// 	   double ef2Param = -1.37350e-01	  ;
// 	   double ef3Param = -1.85806e-01	  ;
// 	   double ef4Param =  2.88660e+00	  ;
// 	   double ef5Param = -2.25131e+01	  ;
// 	   double ef6Param =  6.68356e+01	  ;
// 	   double ef7Param = -6.72583e+01	  ;
//..Erfc
	   double ef0Param =  1.31463e+02  ;
	   double ef1Param =  3.89558e-03  ;
	   double ef2Param =  3.15785e-02  ;
	   double ef3Param = -1.48208e-01  ;
	   double ef4Param =  2.29056e+00  ;
	   double ef5Param = -1.84320e+01  ;
	   double ef6Param =  5.44423e+01  ;
	   double ef7Param = -5.35583e+01  ;
   	   double sef0Param =  1.44466e+00   ;
   	   double sef1Param =  3.90037e-03   ;
	   double sef2Param =  3.15401e-02   ;
     	   double rhoParam  =  0.587	     ;
 	   GooFit::Variable sef11("sef11", 2.087e+00 );
 	   GooFit::Variable sef22("sef22", 3.261e-10 );
 	   GooFit::Variable sef33("sef33", 2.511e-08 );
 	   GooFit::Variable sef12("sef12", 1.531e-05 );
 	   GooFit::Variable sef13("sef13",-1.406e-04 );
 	   GooFit::Variable sef23("sef23",-3.606e-10 );
// 	   double ef0Param =  1.70441e-01	  ;
// 	   double ef1Param =  3.05321e+02	  ;
// 	   double ef2Param = -1.38062e-01	  ;
// 	   double ef3Param = -1.86543e-01	  ;
// 	   double ef4Param =  2.90080e+00	  ;
// 	   double ef5Param = -2.26120e+01	  ;
// 	   double ef6Param =  6.71608e+01	  ;
// 	   double ef7Param = -6.76570e+01	  ;
//    	   double sef0Param =	 3.44552e-03	;
//    	   double sef1Param =	 3.05321e+02	;
// 	   double sef2Param =    3.49278e-03    ;
//      	   double rhoParam  =	 0.895  	;
//   GooFit::Variable* sef11 = new GooFit::Variable("sef11", 1.187e-05 ); 
//   GooFit::Variable* sef22 = new GooFit::Variable("sef22", 1.379e+01 ); 
//   GooFit::Variable* sef33 = new GooFit::Variable("sef33", 1.220e-05 ); 
//   GooFit::Variable* sef12 = new GooFit::Variable("sef12", 1.145e-02 ); 
//   GooFit::Variable* sef13 = new GooFit::Variable("sef13",-1.202e-05 ); 
//   GooFit::Variable* sef23 = new GooFit::Variable("sef23",-1.181e-02 ); 
//	   
// 	   double ef0Param =  1.70820e-01	  ;
// 	   double ef1Param =  3.05391e+02	  ;
// 	   double ef2Param = -1.38397e-01	  ;
// 	   double ef3Param = -1.85847e-01	  ;
// 	   double ef4Param =  2.88704e+00	  ;
// 	   double ef5Param = -2.25124e+01	  ;
// 	   double ef6Param =  6.68205e+01	  ;
// 	   double ef7Param = -6.72272e+01	  ;
//    	   double sef0Param =	 3.47581e-03	;
//    	   double sef1Param =	 3.72946e+00	;
//      	   double rhoParam  =	 0.893  	;
// 	   double sef2Param =    3.52303e-03    ;
// 	   double sef0Param =	 4.39679e-05	;
// 	   double sef1Param =	 1.07946e+00	;
// 	   double rhoParam  =	 -0.433  	;
//   GooFit::Variable* sef11 = new GooFit::Variable("sef11", 1.208e-05 ); 
//   GooFit::Variable* sef22 = new GooFit::Variable("sef22", 1.391e+01 ); 
//   GooFit::Variable* sef33 = new GooFit::Variable("sef33", 1.241e-05 ); 
//   GooFit::Variable* sef12 = new GooFit::Variable("sef12", 1.162e-02 ); 
//   GooFit::Variable* sef13 = new GooFit::Variable("sef13",-1.223e-05 ); 
//   GooFit::Variable* sef23 = new GooFit::Variable("sef23",-1.198e-02 ); 

//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.006-0.35 Els>3 ADAPTIVE
//
 
// 
// I Option i effi fit adaptive 
// 
 
//  
// 	   double ef0Param =    1.91776e-01 	;
// 	   double ef1Param =    3.23179e+02 	;
// 	   double ef2Param =   -1.59890e-01 	;
// 	   double ef3Param =   -1.56484e-01 	;
// 	   double ef4Param =    2.35446e+00 	;
// 	   double ef5Param =   -1.83996e+01 	;
// 	   double ef6Param =    5.29587e+01 	;
// 	   double ef7Param =   -5.05694e+01 	;
//  	   double sef0Param =	4.25355e-03	;
//   	   double sef1Param =	3.55237e+00	;
// 	   double rhoParam  =	0.845		;

// 	   double ef0Param =   2.07891e-02     ;
// 	   double ef1Param =   2.39721e+02     ;
// 	   double ef2Param =   1.08529e-02     ;
// 	   double ef3Param =  -1.55834e-01     ;
// 	   double ef4Param =   2.35394e+00     ;
// 	   double ef5Param =  -1.85136e+01     ;
// 	   double ef6Param =   5.23539e+01     ;
// 	   double ef7Param =  -4.69369e+01     ;
//  	   double sef0Param =  3.03492e-03     ;
//   	   double sef1Param =  2.98088e+00     ;
// 	   double rhoParam  =  0.883	       ;

//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.006-0.35 Els>3 Moro Sigmoid
// 
// 
// 	   double ef0Param =     1.70820e-01  	  ;
// 	   double ef1Param =     3.05391e+02  	  ;
// 	   double ef2Param =    -8.10192e-01  	  ;
// 	   double ef3Param =    -1.08797e+00  	  ;
// 	   double ef4Param =     1.69011e+01  	  ;
// 	   double ef5Param =    -1.31790e+02  	  ;
// 	   double ef6Param =     3.91175e+02  	  ;
// 	   double ef7Param =    -3.93556e+02  	  ;
//   	   double sef0Param =	 4.93457e-03	;
//   	   double sef1Param =	 4.60564e+00	;
// 	   double rhoParam  =	 0.933  	;

//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//GenTrue break 5 sigma 0.006-0.35
//		       double ef0Param =     1.66983e-01    ;
//		       double ef1Param =     3.02531e+02    ;
//		       double ef2Param =    -1.34503e-01    ;
//		       double ef3Param =    -1.88564e-01    ;
//		       double ef4Param =     2.93013e+00    ;
//		       double ef5Param =    -2.28080e+01    ;
//		       double ef6Param =     6.77249e+01    ;
//		       double ef7Param =    -6.82386e+01    ;
////		       double sef0Param =    3.42514e-03    ;
////		       double sef1Param =    3.76065e+00    ;
////		       double rhoParam  =    0.896	    ;
//		       double sef0Param =    4.40192e-05    ;
//		       double sef1Param =    3.02532e+02    ;
// 		     double rhoParam  =    -0.434	  ;
//       	     double ef0Param = 1.66895e-01	 ;
//       	     double ef1Param = 3.02444e+02	 ;
//       	     double ef2Param =-1.34418e-01	 ;
//       	     double ef3Param =-1.88642e-01	 ;
//       	     double ef4Param = 2.93237e+00	 ;
//       	     double ef5Param =-2.28278e+01	 ;
//       	     double ef6Param = 6.77957e+01	 ;
//       	     double ef7Param =-6.83260e+01	 ;
// //////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////

//  GooFit::Variable* ef0 = new GooFit::Variable("ef0",  5.06489e-02); 
//  GooFit::Variable* uno = new GooFit::Variable("uno",  1); 
//   GooFit::Variable* ef0 = new GooFit::Variable("ef0",  ef0Param,ef0Param- 3.41501e-03,ef0Param+ 3.41501e-03); 
//   GooFit::Variable* ef1 = new GooFit::Variable("ef1",  ef1Param,ef1Param-3.67208e+00,ef1Param+3.67208e+00); 
 double NSigEffi0  =    0.5;
 double NSigEffi1  =    1.0;
 double NSigEffi2  =    0.5;
  GooFit::Variable efx("efx",  ef0Param,ef0Param - NSigEffi0*sef0Param,ef0Param + NSigEffi0*sef0Param); 
  GooFit::Variable efy("efy",  ef1Param,ef1Param - NSigEffi1*sef1Param,ef1Param + NSigEffi1*sef1Param); 
  GooFit::Variable efz("efz",  ef2Param,ef2Param - NSigEffi2*sef2Param,ef2Param + NSigEffi2*sef2Param); 
//  GooFit::Variable* efx = new GooFit::Variable("efx",  ef0Param); 
//  GooFit::Variable* efy = new GooFit::Variable("efy",  ef1Param); 
  GooFit::Variable ef0("ef0",  ef0Param); 
  GooFit::Variable ef1("ef1",  ef1Param); 
  GooFit::Variable ef2("ef2",  ef2Param); 
  GooFit::Variable ef3("ef3",  ef3Param); 
  GooFit::Variable ef4("ef4",  ef4Param);  
  GooFit::Variable ef5("ef5",  ef5Param); 
  GooFit::Variable ef6("ef6",  ef6Param); 
  GooFit::Variable ef7("ef7",  ef7Param); 

  GooFit::Variable sef0("sef0",  sef0Param); 
  GooFit::Variable sef1("sef1",  sef1Param); 
  GooFit::Variable rho ("rho" ,  rhoParam); 


//  coeffEffi.push_back(uno);
//   vector<GooFit::Variable*> coeffEffi;
//   coeffEffi.push_back(ef0);
//   coeffEffi.push_back(ef1);
//   coeffEffi.push_back(ef2);
//  coeffEffi.push_back(ef3);

 
//   PolyEffiPdf* Effi = new PolyEffiPdf("Effi", xcTau, coeffEffi);
// PolynomialPdf* Effi = new PolynomialPdf("Effi", xcTau, coeffEffi);

//  ErfcPolyPdf  *Effi = new ErfcPolyPdf("Effi", xcTau, ef0,ef1,ef2,ef3,ef4);
//      SigmoidBpMoroPdf  *Effi = new SigmoidBpMoroPdf("Effi", xcTau, efx,efy,ef2,ef3,ef4,ef5,ef6,ef7);
//      SigmoidBpPdf  *Effi    = new SigmoidBpPdf("Effi", xcTau, efx,efy,efz,ef3,ef4,ef5,ef6,ef7);
//      SigmoidBpPdf  *Effi = new SigmoidBpPdf("Effi", xcTau, efx,efy,ef2,ef3,ef4,ef5,ef6,ef7);
//      SigmoidBpPdf  *Effi = new SigmoidBpPdf("Effi", xcTau, ef0,ef1,ef2,ef3,ef4,ef5,ef6,ef7);
//      SigmoidGausPdf  *Effi = new SigmoidGausPdf("Effi", xcTau, ef0,ef1,ef2,ef3,ef4,ef5,ef6);

//      SigmoidBpPdf  *EffiFIX = new SigmoidBpPdf("EffiFIX", xcTau, ef0,ef1,ef2,ef3,ef4,ef5,ef6,ef7);
      ErfEffiBpPdf  *EffiFIX = new ErfEffiBpPdf("EffiFIX", xcTau, ef0,ef1,ef2,ef3,ef4,ef5,ef6,ef7);


     GooFit::Variable XMinV ("XMinV"  ,XMin );
     GooFit::Variable XMaxV ("XMaxV"  ,XMax );
     GooFit::Variable SXMinV("SXMinV" ,SXMin);
     GooFit::Variable SXMaxV("SXMaxV" ,SXMax);
//  Effi->setParameterConstantness(true); 
//  pdfFitBckg1 ->addSpecialMask(PdfBase::ForceSeparateNorm); 
//  Effi        ->addSpecialMask(PdfBase::ForceSeparateNorm); 
   
//    ExpGausProdBPdf* DecayBp	= new ExpGausProdBPdf("DecayBp"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign, meanGaussianErrSign,tauErrSign1,
//    XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* DecayBp1	= new ExpGausProdBPdf("DecayBp1"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign1, meanGaussianErrSign1,tauErrSign1,
      XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdEffiBPdf* DecayBp1	= new ExpGausProdEffiBPdf("DecayBp1"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign, meanGaussianErrSign,tauErrSign1,
//  XMinV,XMaxV,SXMinV,SXMaxV);
//     ExpGausProdBPdf* DecayBp2	= new ExpGausProdBPdf("DecayBp2"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign2, meanGaussianErrSign2,tauErrSign2,
//     XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* DecayBp2	= new ExpGausProdBPdf("DecayBp2"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign2, meanGaussianErrSign2,tauErrSign1,
      XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* DecayBp3 = new ExpGausProdBPdf("DecayBp3"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign3, meanGaussianErrSign3,tauErrSign1,
      XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdEffiBPdf* DecayBp2	= new ExpGausProdEffiBPdf("DecayBp2"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign2, meanGaussianErrSign2,tauErrSign2,
//  XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdEffiBPdf* pdfFitBckg1 = new ExpGausProdEffiBPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg, tauSB1, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
//  XMinV,XMaxV,SXMinV,SXMaxV);
//        ExpGausProdBPdf* pdfFitBckg1 = new ExpGausProdBPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg, tauSB1, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
//        XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* pdfFitBckg1 = new ExpGausProdBPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg2, tauSB1, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
      XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* pdfFitBckg2 = new ExpGausProdBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg2, tauSB2, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
      XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* pdfFitBckg3 = new ExpGausProdBPdf("pdfFitBckg3", xcTau, xScTau, meanResBckg2, tauSB3, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
      XMinV,XMaxV,SXMinV,SXMaxV);
//        ExpGausProdBPdf* pdfFitBckg3 = new ExpGausProdBPdf("pdfFitBckg3", xcTau, xScTau, meanResBckg3, tauSB3, sigmaGaussianErrorBckg3, meanGaussianErrBckg3,tauErrBckg3,
//        XMinV,XMaxV,SXMinV,SXMaxV);
 // ExpGausProdEffiBPdf* pdfFitBckg2 = new ExpGausProdEffiBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
 // XMinV,XMaxV,SXMinV,SXMaxV);
//       ExpGausProdBPdf* pdfFitBckg2 = new ExpGausProdBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2, sigmaGaussianErrorBckg2, meanGaussianErrBckg2,tauErrBckg2,
//       XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdBPdf* pdfFitBckg2 = new ExpGausProdBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg2,
//  XMinV,XMaxV,SXMinV,SXMaxV);

// ExpPdf* DecayBp     = new ExpPdf("DecayBp"    ,  xcTau, cTau  );
// ExpPdf* pdfFitBckg1 = new ExpPdf("pdfFitBckg1", xcTau, tauSB1);
// ExpPdf* pdfFitBckg2 = new ExpPdf("pdfFitBckg2", xcTau, tauSB2);


//  ExpGausPEEPdf* DecayBp     = new ExpGausPEEPdf("DecayBp"    , xcTau, xScTau, meanResSign   , cTau  );
//  ExpGausPEEPdf* pdfFitBckg1 = new ExpGausPEEPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg, tauSB1);
//  ExpGausPEEPdf* pdfFitBckg2 = new ExpGausPEEPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2);

  
//  ExpGausPdf* DecayBp	    = new ExpGausPdf("DecayBp"	, xcTau, meanResSign   , sigmaRes, cTau  );
//  ExpGausPdf* pdfFitBckg1 = new ExpGausPdf("pdfFitBckg1", xcTau, meanResBckg, sigmaResBckg, tauSB1);
//  ExpGausPdf* pdfFitBckg2 = new ExpGausPdf("pdfFitBckg2", xcTau, meanResBckg, sigmaResBckg, tauSB2);

//    std::vector<PdfBase*> compspdfFitBp1;
//    compspdfFitBp1.push_back(DecayBp1);
//    compspdfFitBp1.push_back(Effi);
//    ProdPdf* pdfFitBp1	 = new ProdPdf("pdfFitBp1"  , compspdfFitBp1);
// 
//    std::vector<PdfBase*> compspdfFitBp2;
//    compspdfFitBp2.push_back(DecayBp2);
//    compspdfFitBp2.push_back(Effi);
//    ProdPdf* pdfFitBp2	 = new ProdPdf("pdfFitBp2"  , compspdfFitBp2);
// 
//   std::vector<PdfBase*> compspdfDecayBpAdd;
//   compspdfDecayBpAdd.push_back(pdfFitBp1);
//   compspdfDecayBpAdd.push_back(pdfFitBp2);
//   GooFit::AddPdf *pdfFitBp = new GooFit::AddPdf("pdfFitBp", weightsSignalMass, compspdfDecayBpAdd);
// 
   std::vector<GooFit::Variable> weightsSignalTau;
   weightsSignalTau.push_back(wt1);
//   weightsSignalTau.push_back(wt2); 
   std::vector<PdfBase*> compspdfDecayBpAdd;
   compspdfDecayBpAdd.push_back(DecayBp1);
   compspdfDecayBpAdd.push_back(DecayBp2);
//   compspdfDecayBpAdd.push_back(DecayBp3);
 GooFit::AddPdf DecayBp("DecayBp", weightsSignalMass, compspdfDecayBpAdd);
//     GooFit::AddPdf DecayBp("DecayBp", weightsSignalTau, compspdfDecayBpAdd);
//  GooFit::AddPdf* pdfFitBp =  new GooFit::AddPdf("pdfFitBp", weightsSignalMass, compspdfDecayBpAdd);
  
 
//  DecayBp.addSpecialMask(PdfBase::ForceSeparateNorm); 
 
 
 
    std::vector<PdfBase*> compspdfFitBp;
    compspdfFitBp.push_back(&DecayBp);
//    compspdfFitBp.push_back(DecayBp1);
    compspdfFitBp.push_back(EffiFIX);
    ProdPdf* pdfFitBp	 = new ProdPdf("pdfFitBp"  , compspdfFitBp);

  GooFit::Variable b1("b1",1.21140e-01,0., 1.);
  GooFit::Variable b2("b2",5.38694e-02,0., 1.);
  std::vector<GooFit::Variable> weightspdfFitBckg;
  weightspdfFitBckg.push_back(b1);
//  weightspdfFitBckg.push_back(b2);

  
  std::vector<PdfBase*> compspdfFitBckgAdd;
  compspdfFitBckgAdd.push_back(pdfFitBckg1);
  compspdfFitBckgAdd.push_back(pdfFitBckg2);
//  compspdfFitBckgAdd.push_back(pdfFitBckg3);
//  GooFit::AddPdf pdfFitBckgAdd("pdfFitBckgAdd", weightspdfFitBckg, compspdfFitBckgAdd);
//  GooFit::AddPdf pdfFitBckgTmp("pdfFitBckgTmp",weightsBckgMass , compspdfFitBckgAdd);
  GooFit::AddPdf pdfFitBckgTmp("pdfFitBckgTmp", weightspdfFitBckg, compspdfFitBckgAdd);
//      GooFit::AddPdf pdfFitBckg("pdfFitBckg", weightspdfFitBckg, compspdfFitBckgAdd);
  
   std::vector<PdfBase*> compspdfFitBpBckg;
   compspdfFitBpBckg.push_back(&pdfFitBckgTmp);
//   compspdfFitBpBckg.push_back(pdfFitBckg1);
   compspdfFitBpBckg.push_back(EffiFIX);
   ProdPdf pdfFitBckg("pdfFitBckg"  , compspdfFitBpBckg);
   
////////////////////////////////////////////////////   
////////////////////////////////////////////////////   
// bivariate gaussian constraint
////////////////////////////////////////////////////   
////////////////////////////////////////////////////   

// BivarGaussianConstrPdf* EffiConstr = new BivarGaussianConstrPdf("EffiConstr",xcTau, efx,efy, ef0,sef0, ef1,sef1, rho);   
// TrivarGaussianConstrPdf* EffiConstr = new TrivarGaussianConstrPdf("EffiConstr",xcTau, efx,efy,efz,ef0,ef1,ef2,sef11,sef22,sef33,sef12,sef13,sef23);   

////////////////////////////////////////////////////   
////////////////////////////////////////////////////   
// bivariate gaussian constraint
////////////////////////////////////////////////////   
////////////////////////////////////////////////////   
   
   
 
//  std::vector<PdfBase*> compspdfFitBckg;
//  compspdfFitBckg.push_back(pdfFitBckg1);
//  compspdfFitBckg.push_back(&pdfFitBckgAdd);
//  compspdfFitBckg.push_back(Effi);
 
//  ProdPdf* pdfFitBckg   = new ProdPdf("pdfFitBckg", compspdfFitBckg);

// Res Models   
//  ExpGausPEESigmaBPdf* ExpGauSign = new ExpGausPEESigmaBPdf("ExpGauSig" , xScTau,  sigmaGaussianErrorSign, meanGaussianErrSign,tauErrSign1,
//    SXMinV,SXMaxV);
//  ExpGausPEESigmaBPdf* ExpGauBckg = new ExpGausPEESigmaBPdf("ExpGauBckg", xScTau,  sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
//    SXMinV,SXMaxV);
//  ExpGausPEESigmaBPdf* ExpGauBckg2 = new ExpGausPEESigmaBPdf("ExpGauBckg2", xScTau,  sigmaGaussianErrorBckg2, meanGaussianErrBckg2,tauErrBckg2,
//    SXMinV,SXMaxV);


//  GooPdf* LandauErrorSign = new LandauPdf("LandauErrorSign", xScTau, meanLandauErrSign, sigmaLandauErrorSign);
//  GooPdf* LandauErrorBckg = new LandauPdf("LandauErrorBckg", xScTau, meanLandauErrBckg, sigmaLandauErrorBckg);

//  GooPdf* GaussianErrorSign = new GaussianPdf("GaussianErrorSign", xScTau, meanGaussianErrSign, sigmaGaussianErrorSign);
//  GooPdf* GaussianErrorBckg = new GaussianPdf("GaussianErrorBckg", xScTau, meanGaussianErrBckg1, sigmaGaussianErrorBckg);

//  GooPdf* BifurGErrorSign = new BifurGaussPdf("BifurGErrorSign", xScTau, meanBifurGErrSign, sigmaLBifurGErrSign,sigmaRBifurGErrSign);
//  GooPdf* BifurGErrorBckg = new BifurGaussPdf("BifurGErrorBckg", xScTau, meanBifurGErrBckg, sigmaLBifurGErrBckg,sigmaRBifurGErrBckg);

//  ExpGausPdf* ExpGauSign = new ExpGausPdf("ExpGauSig" , xScTau, meanGaussianErrSign, sigmaGaussianErrorSign, tauErrSign);
//  ExpGausPdf* ExpGauBckg = new ExpGausPdf("ExpGauBckg", xScTau, meanGaussianErrBckg1, sigmaGaussianErrorBckg1, tauErrBckg);

///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
//
// 2DFit
//  
///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
  

  std::vector<PdfBase*> compsSignalLife;
  compsSignalLife.push_back(&signalMass);
  compsSignalLife.push_back(pdfFitBp);
//  compsSignalLife.push_back( GaussianErrorSign);
  //compsSignalLife.push_back(ExpGauSign);
//  compsSignalLife.push_back( LandauErrorSign);
  ///compsSignalLife.push_back(BifurGErrorSign);
//  compsSignalLife.push_back(DecayBp);

  std::vector<PdfBase*> compsBckgLife;
  compsBckgLife.push_back(&bckgMass);
  compsBckgLife.push_back(&pdfFitBckg);
//  compsBckgLife.push_back(pdfFitBckg1);
//  compsSignalLife.push_back( GaussianErrorBckg);
  //compsSignalLife.push_back(ExpGauBckg);
//  compsSignalLife.push_back( LandauErrorBckg);
  //compsSignalLife.push_back(BifurGErrorBckg);
//compsBckgLife.push_back(&pdfFitBckgAdd);

  ProdPdf* signalLife = new ProdPdf("signalLife", compsSignalLife);
  ProdPdf* bckgLife   = new ProdPdf("bckgLife  ", compsBckgLife);

  std::vector<GooFit::Variable> weightsYield;
  weightsYield.push_back(*signalYield);
  weightsYield.push_back(*bckgYield);
  

  std::vector<PdfBase*> compsModel;
  compsModel.push_back(signalLife);
//  compsModel.push_back(gaussBckgBp);
  compsModel.push_back(bckgLife);
  
  
//  compsModel.push_back(poly);
//  compsModel.push_back(argus);
  GooFit::AddPdf model("model", weightsYield, compsModel); 
//  GooFit::AddPdf model1("model1", weightsYield, compsModel); 

//  GooFit::Variable* uno = new GooFit::Variable("uno",0.5);
//  GooFit::Variable* L   = new GooFit::Variable("L",-100.,-100.,-100.);
//  std::vector<PdfBase*> compsModelConstr;
//    compsModelConstr.push_back(&model1);
//    compsModelConstr.push_back(EffiConstr);
//    compsModelConstr.push_back(signalLife);
//    compsModelConstr.push_back(bckgLife);
//  compsModelConstr.push_back(EffiConstr);

//  std::vector<GooFit::Variable*> weightsC;
//   weightsC.push_back(signalYield);
//   weightsC.push_back(bckgYield);
//  weightsC.push_back(uno);
//  weightsC.push_back(uno);

//  GooFit::AddPdf model("model", weightsC , compsModelConstr); 
//  ProdPdf model("model",  compsModelConstr); 


//  model.addSpecialMask(PdfBase::ForceCommonNorm) ;

//
// These are used for Plots....
//
  std::vector<PdfBase*> compsMass;
  compsMass.push_back(&signalMass);
  compsMass.push_back(&bckgMass);
  GooFit::AddPdf modelMass("modelMass", weightsYield, compsMass); 
  
  std::vector<PdfBase*> compscTau;
  compscTau.push_back(pdfFitBp);
  compscTau.push_back(&pdfFitBckg);
  GooFit::AddPdf model_cTau("model_cTau", weightsYield, compscTau); 
  
//  std::vector<PdfBase*> compsSTau;
//  compsSTau.push_back(LandauErrorSign);
//  compsSTau.push_back(LandauErrorBckg);
//  compsSTau.push_back(BifurGErrorSign);
//  compsSTau.push_back(BifurGErrorBckg);
//  compsSTau.push_back(GaussianErrorSign);
//  compsSTau.push_back(GaussianErrorBckg);
//  compsSTau.push_back(ExpGauSign);
//  compsSTau.push_back(ExpGauBckg);
//  GooFit::AddPdf model_STau("model_cTau", weightsYield, compsSTau); 
  
  

//
// Data
//
  std::vector<GooFit::Observable> dataVec;
  
  dataVec.push_back(xMass);
  dataVec.push_back(xcTau);
  dataVec.push_back(xScTau);
  UnbinnedDataSet* dataLife = new GooFit::UnbinnedDataSet(dataVec);
//
  if (!InputFile)
   {
     cout<<"File:"<<InputFileName<<" not found!!!"<<endl;
    exit(1);
   }
   InputFile->ls();
   
   TTree *TauBpTree    = (TTree*)InputFile->Get(InputTauBpTreeName);
   if(!TauBpTree ){
     cout<<"TTree cTau Data: "<< InputTauBpTreeName <<" not found!!!"<<endl;
     exit(1);
   }else{
     cout<<"TTree cTau Data: "<< InputTauBpTreeName <<" OK FOUND!!!"<<endl;
   }  
    
   TauBpTree->SetBranchAddress("xBpMass",&xBpMass);
//   TauBpTree->SetBranchAddress("xBpTau" ,&xBpTau);
   TauBpTree->SetBranchAddress("xBpcTau",&xBpcTau);
   TauBpTree->SetBranchAddress("xSBpcTau",&xSBpcTau);
   int nentries = (int)TauBpTree->GetEntries();
   
   for (Int_t i=0;i<nentries;i++) { 
    TauBpTree->GetEntry(i);
    if(xBpcTau>=XMin&&xBpcTau<=XMax&&xSBpcTau>=SXMin&&xSBpcTau<=SXMax){
     if(xBpMass>=XMinSign&&xBpMass<=XMaxSign){
      xMass.setValue(xBpMass)  ;
      xcTau.setValue(xBpcTau)   ;
      xScTau.setValue(xSBpcTau); 
      dataLife->addEvent();
      HxMass.Fill(xBpMass);
      HxcTau.Fill(xBpcTau);
      HxScTau.Fill(xSBpcTau);
     } 
     if(xBpMass>XMinSBL&&xBpMass<XMaxSBL){
        HxcTauSB.Fill(xBpcTau);
        HxScTauSB.Fill(xSBpcTau);
     } 
     if(xBpMass>XMinSBR&&xBpMass<XMaxSBR){ 
        HxcTauSB.Fill(xBpcTau);
        HxScTauSB.Fill(xSBpcTau);
     }  
    } 
   }
   char TXT [200];
   sprintf(TXT,"Mass        Entries = %7f",HxMass.GetEntries());
   cout<<"***********************************"<<endl;
   cout<<"***********************************\n"<<endl;
   cout<<"TauBpTree   Entries = "<<nentries<<endl;
   cout<<"Mass        Entries = "<<HxMass.GetEntries()<<endl;
   cout<<TXT<<endl;
   cout<<"SideBand    Entries = "<<HxcTauSB.GetEntries()<<endl;
   cout<<"\n***********************************"<<endl;
   cout<<"***********************************"<<endl;
//================================================================================
//================================================================================
///FIT
//================================================================================
//================================================================================
//  double arglis[10];
//  int ierflg= 0;
  model.setData(dataLife);
//  FitManager fitter(&modelC);
  GooFit::FitManagerMinuit1 fitter(&model);
  fitter.setMaxCalls(12000);
  cout<<"                  ===*** Start Fit ***=== "<<endl;
  cout<<"                  ===*** Start Fit ***=== "<<endl;
  cout<<"                  ===*** Start Fit ***=== "<<endl;
//  fitter.setupMinuit();
  Minuit1 * Minuit = fitter.getMinuitObject();
  Minuit->SetPrintLevel(1);
  Minuit->SetErrorDef(1);
//  Minuit->SetErrorDef(0.5);
  Double_t arglist[2]; int err = 0;
  arglist[0]= 12000; // maximum iterations
  arglist[1]= 1.0; 
  Minuit->mnexcm("MIGRAD",arglist,2,err);
  Minuit->mnexcm("HESSE",arglist, 0,err);

//  Minuit->SetErrorDef(0.5);

//   double arglis[10];
//   arglis[0]=8;
//   int ierflg= 0;
//   Minuit->mnexcm("FIX ",arglis,1,ierflg);
//   fitter.runCommand("MIGRAD");
//   for (int i=19;i<23;i++){
//    arglis[0]=i;
//    Minuit->mnexcm("FIX ",arglis,1,ierflg);
//   } 
//   fitter.runCommand("MIGRAD");
//   for (int i=1;i<37;i++){
//    arglis[0]=i;
//    Minuit->mnexcm("FIX ",arglis,1,ierflg);
//   }
//   for (int i=19;i<23;i++){
//    arglis[0]=i;
//    Minuit->mnexcm("REL ",arglis,1,ierflg); 
//   }
//   fitter.runCommand("MIGRAD");
//   for (int i=19;i<20;i++){
//    arglis[0]=i;
//    Minuit->mnexcm("REL ",arglis,1,ierflg); 
//   }
//      Minuit->FixParameter(0);
//      Minuit->FixParameter(1);
//      Minuit->FixParameter(2);
//      Minuit->FixParameter(3);
//      Minuit->FixParameter(4);
//      Minuit->FixParameter(5);
//      Minuit->FixParameter(6);

// 
//       Minuit->FixParameter(7);
// 
/* 
       Minuit->FixParameter(35);
       Minuit->FixParameter(36);
       Minuit->FixParameter(37);
       Minuit->FixParameter(38);
       Minuit->FixParameter(39);
       Minuit->FixParameter(40);
       Minuit->FixParameter(41);
       Minuit->FixParameter(42);
       Minuit->FixParameter(43);
       Minuit->FixParameter(44);
       Minuit->FixParameter(45);
       Minuit->FixParameter(46);
       Minuit->FixParameter(47);

 */
//      fitter.runCommand("MIGRAD");
   
//      Minuit->Release(7);
      
      
 //       Minuit->Release(36);
//        Minuit->Release(38);
//          Minuit->Release(39);
//          Minuit->Release(40);
//        Minuit->Release(41);
//        Minuit->Release(42);
//        Minuit->Release(43);
//        Minuit->Release(44);
//        Minuit->Release(45);
//        Minuit->Release(46);
// 
//
//       Minuit->Release(7);
//       fitter.runCommand("MIGRAD");

//      Minuit->FixParameter(18);
//      Minuit->FixParameter(19);
//      Minuit->FixParameter(20);
//      Minuit->FixParameter(21);
//Minuit->FixParameter(15);
//Minuit->FixParameter(16);
//Minuit->FixParameter(17);
//Minuit->FixParameter(18);
//     fitter.runCommand("MIGRAD"); 
//    for (int i=0;i<37;i++){
//     arglis[0]=i;
//     Minuit->mnexcm("FIX ",arglis,1,ierflg);
//    }
/*  for (int i=19;i<23;i++){
   arglis[0]=i;
   Minuit->mnexcm("REL ",arglis,1,ierflg);
  }
    arglis[0]=8;
      Minuit->mnexcm("REL ",arglis,1,ierflg);
*/
//    arglis[0]=8;
//    Minuit->mnexcm("REL ",arglis,1,ierflg);
//    arglis[0]=10;
//    Minuit->mnexcm("REL ",arglis,1,ierflg);
//    arglis[0]=16;
//    Minuit->mnexcm("REL ",arglis,1,ierflg);
//    arglis[0]=17;
//    Minuit->mnexcm("REL ",arglis,1,ierflg);
//     arglis[0]=34;
//     Minuit->mnexcm("REL ",arglis,1,ierflg);

//     Minuit->FixParameter(44);
//     Minuit->FixParameter(45);
//     Minuit->FixParameter(46);
//     Minuit->FixParameter(47);
    //fitter.runCommand("MIGRAD");
//    fitter.runMigrad(0.3);
 
//  fitter.runCommand("MINOS");
    //fitter.runCommand("HESSE");
//  fitter.fit();   
//  Minuit->SetPrintLevel(1);
//  Minuit->mnmigr();
//  Minuit->mnhess();
//  Minuit->mnmigr();
  std::vector<Variable> var; 
  double tmp_value, tmp_error;
  for(Variable &var : Minuit->getVaraibles()) {
      Minuit->GetParameter(var.getFitterIndex(), tmp_value, tmp_error);
      var.setValue(tmp_value);
      var.setError(tmp_error);
  }
//  fitter.getMinuitValues(); 
  cout<<"		   ===***  End  Fit ***=== "<<endl;
  cout<<"		   ===***  End  Fit ***=== "<<endl;
  cout<<"		   ===***  End  Fit ***=== "<<endl;
  
//================================================================================
//================================================================================
///FIT
//================================================================================
//================================================================================

//================================================================================
///PLOT

//XHScale=10;
// Mass
  UnbinnedDataSet gridMass(xMass);
  double totalDataMass = 0; 
  double NStepMass = XHScale*xMass.getNumBins();
  for (int i = 0; i < NStepMass; ++i) {
    double step = (xMass.getUpperLimit() - xMass.getLowerLimit())/NStepMass;
    xMass.setValue(xMass.getLowerLimit() + (i + 0.5) * step);
    gridMass.addEvent(); 
   totalDataMass++; 
  }

  modelMass.setData(&gridMass);
  std::vector<std::vector<double> > pdfValsMass = modelMass.getCompProbsAtDataPoints();
//  modelMass.getCompProbsAtDataPoints(pdfValsMass); 
  double totalPdfMass = 0; 
  for (int i = 0; i < gridMass.getNumEvents(); ++i) {
    gridMass.loadEvent(i); 
    pdfHist.Fill(xMass.getValue(), pdfValsMass[0][i]);
    sigHist.Fill(xMass.getValue(), pdfValsMass[1][i]);
    bkgHist.Fill(xMass.getValue(), pdfValsMass[2][i]);
    totalPdfMass += pdfValsMass[0][i]; 
  }
  
  
  pdfHist.Scale((signalYield->getValue()+bckgYield->getValue())/pdfHist.Integral()*XHScale);
  sigHist.Scale(signalYield->getValue()/sigHist.Integral()*XHScale);
  bkgHist.Scale(bckgYield->getValue()/bkgHist.Integral()*XHScale);
  std::cout<<"Signal Yield = "<< signalYield->getValue()<<std::endl;
  std::cout<<"Bckg   Yield = "<< bckgYield->getValue()<<std::endl;
  std::cout<<"(SB    Yield  = "<<HxcTauSB.GetEntries() <<")"<<std::endl;
  std::cout<<"Tot   Yield  = "<< signalYield->getValue()+bckgYield->getValue()<<std::endl;
//--------------------------------------------------  
// Tau


//XHScale=1;
  int NIntegral = 1;
  

//  vector<GooFit::Variable*> dataPlot;
//  dataPlot.push_back(xcTau);
//  dataPlot.push_back(xScTau);

  std::vector<GooFit::Observable> dataPlot2D;
  dataPlot2D.push_back(xcTau);
  dataPlot2D.push_back(xScTau);

//  vector<GooFit::Variable*> dataPlotS;
//  dataPlotS.push_back(xScTau);
  
  
//  UnbinnedDataSet grid_cTau(dataPlot);
  UnbinnedDataSet grid_cTau2D(dataPlot2D);
//  UnbinnedDataSet grid_STau(dataPlotS);
  
//  bool first = true;
//  UnbinnedDataSet grid_cTau(xcTau);
//  double totalData_cTau = 0; 
  double NStep  = XHScale*xcTau.getNumBins();
//  double NSStep   = XHScale*xcTau->getNumBins();
  double NSStep2D = NIntegral*XHScale*xScTau.getNumBins();
  double step  = (xcTau.getUpperLimit() - xcTau.getLowerLimit())/NStep;
//  double sstep = (xScTau.getUpperLimit() - xScTau.getLowerLimit())/NSStep;
  double sstep2D = (xScTau.getUpperLimit() - xScTau.getLowerLimit())/NSStep2D;
  for (int i = 0; i < NStep; ++i) {
    xcTau.setValue(xcTau.getLowerLimit()  + (i + 0.5) * step);
//    grid_cTau.addEvent(); 
//    totalData_cTau++; 
//    xScTau.getValue() = xScTau.getLowerLimit() + (i + 0.5) * sstep;
//    cout<<"X = "<<xcTau->getValue()<<" sx = "<<xScTau.getValue()<<endl;
//    grid_cTau.addEvent(); 
//    xcTau2D->getValue()  = xcTau2D ->getLowerLimit() + (i + 0.5) * step;
//   cout<<"======================================     \n"<<endl;
//   cout<<"======================================     \n"<<endl;
//   cout<<"======================================     \n"<<endl;
    for (int ii = 0; ii < NSStep2D; ++ii) {
     xScTau.setValue(xScTau.getLowerLimit() + (ii + 0.5) * sstep2D);
//     xScTau2D->getValue() = xScTau2D->getLowerLimit() + (ii + 0.5) * sstep2D;
//    cout<<"X = "<<xcTau->getValue()<<" sx = "<<xScTau.getValue()<<endl;
     grid_cTau2D.addEvent(); 
//     if (first) grid_STau.addEvent();
    }
//    first = false;
  }

//  model_cTau.setData(&grid_cTau);
//  vector<vector<double> > pdfVals_cTau;
//  model_cTau.getCompProbsAtDataPoints(pdfVals_cTau); 
//  double totalPdf_cTau = 0; 
//   for (int i = 0; i < grid_cTau.getNumEvents(); ++i) {
//     grid_cTau.loadEvent(i); 
//     pdf_cTau_Hist.Fill(xcTau->getValue() , pdfVals_cTau[0][i]);
//     sig_cTau_Hist.Fill(xcTau->getValue() , pdfVals_cTau[1][i]);
//     bkg_cTau_Hist.Fill(xcTau->getValue() , pdfVals_cTau[2][i]);
//     totalPdf_cTau += pdfVals_cTau[0][i]; 
//   }

//  double pdf_cTau_Integral2D = 0;
//  double sig_cTau_Integral2D = 0;
//  double bkg_cTau_Integral2D = 0;
//model.setData(&grid_cTau2D);
//vector<vector<double> > pdfVals_cTau2D;
//model.getCompProbsAtDataPoints(pdfVals_cTau2D);
     model_cTau.setData(&grid_cTau2D);
     vector<vector<double> > pdfVals_cTau2D = model_cTau.getCompProbsAtDataPoints();
//     model_cTau.getCompProbsAtDataPoints(pdfVals_cTau2D);
  for (int i = 0; i < grid_cTau2D.getNumEvents(); ++i) {
    grid_cTau2D.loadEvent(i); 
    pdf_cTauSTau_Hist2D.Fill(xcTau.getValue() ,xScTau.getValue() , pdfVals_cTau2D[0][i]);
    sig_cTauSTau_Hist2D.Fill(xcTau.getValue() ,xScTau.getValue() , pdfVals_cTau2D[1][i]);
    bkg_cTauSTau_Hist2D.Fill(xcTau.getValue() ,xScTau.getValue() , pdfVals_cTau2D[2][i]);
//     if (i%int(NSStep2D) == 1 && i>0){
//      pdf_cTau_Hist2D.Fill(xcTau->getValue() , pdf_cTau_Integral2D/step);
//      sig_cTau_Hist2D.Fill(xcTau->getValue() , sig_cTau_Integral2D/step);
//      bkg_cTau_Hist2D.Fill(xcTau->getValue() , bkg_cTau_Integral2D/step);
// //     cout<<"Int = "<<bkg_cTau_Integral2D<<endl;
//      pdf_cTau_Integral2D=0;
//      sig_cTau_Integral2D=0;
//      bkg_cTau_Integral2D=0;
// //     exit(0);
//     }else{
// //     cout<<"X = "<<xcTau->getValue()<<" sx = "<<xScTau.getValue()<<" NStep2D = "<<NStep2D<<endl;
//      pdf_cTau_Integral2D =+ pdfVals_cTau2D[0][i]*sstep2D; 
//      sig_cTau_Integral2D =+ pdfVals_cTau2D[1][i]*sstep2D; 
//      bkg_cTau_Integral2D =+ pdfVals_cTau2D[2][i]*sstep2D;
// //     cout<<"Int = "<<pdfVals_cTau2D[2][i]<<endl;
//     } 
  }
  
  TH1D * pdf_cTauSTau_X = pdf_cTauSTau_Hist2D.ProjectionX("pdf_cTauSTau_X");
  TH1D * pdf_cTauSTau_Y = pdf_cTauSTau_Hist2D.ProjectionY("pdf_cTauSTau_Y");

  TH1D * sig_cTauSTau_X = sig_cTauSTau_Hist2D.ProjectionX("sig_cTauSTau_X");
  TH1D * sig_cTauSTau_Y = sig_cTauSTau_Hist2D.ProjectionY("sig_cTauSTau_Y");

  TH1D * bkg_cTauSTau_X = bkg_cTauSTau_Hist2D.ProjectionX("bkg_cTauSTau_X");
  TH1D * bkg_cTauSTau_Y = bkg_cTauSTau_Hist2D.ProjectionY("bkg_cTauSTau_Y");

/*   vector<GooFit::Variable*> dataSPlot;
  dataSPlot.push_back(xScTau);
  dataSPlot.push_back(xcTau);
  UnbinnedDataSet grid_STau(dataSPlot);
//  UnbinnedDataSet grid_cTau(xcTau);
  double totalData_STau = 0; 
  NStep  = XHScale*xScTau.getNumBins();
  double NSStep = XHScale*xcTau->getNumBins();
  for (int i = 0; i < NSStep; ++i) {
    totalData_STau++; 
    double sstep = (xScTau.getUpperLimit() - xScTau.getLowerLimit())/NSStep;
    xScTau.getValue() = xScTau.getLowerLimit() + (i + 0.5) * sstep;
    grid_STau.addEvent(); 
  }
 */

//
// STau
//
//   model_STau.setData(&grid_STau);
//   vector<vector<double> > pdfVals_STau;
//   model_STau.getCompProbsAtDataPoints(pdfVals_STau); 
//   double totalPdf_STau = 0;  
//   for (int i = 0; i < grid_STau.getNumEvents(); ++i) {
//     grid_STau.loadEvent(i); 
//     pdf_STau_Hist.Fill(xScTau.getValue(), pdfVals_STau[0][i]);
//     sig_STau_Hist.Fill(xScTau.getValue(), pdfVals_STau[1][i]);
//     bkg_STau_Hist.Fill(xScTau.getValue(), pdfVals_STau[2][i]);
//     totalPdf_STau += pdfVals_STau[0][i]; 
//   }
  
//
// Models plot  
//   pdf_cTau_Hist.Scale((signalYield->getValue()+bckgYield->getValue())/pdf_cTau_Hist.Integral()*XHScale);
//   sig_cTau_Hist.Scale(signalYield->getValue()/sig_cTau_Hist.Integral()*XHScale);
//   bkg_cTau_Hist.Scale(HxcTauSB.GetEntries()/bkg_cTau_Hist.Integral()*XHScale);
// 
//   pdf_cTau_Hist2D.Scale((signalYield->getValue()+bckgYield->getValue())/pdf_cTau_Hist2D.Integral()*XHScale);
//   sig_cTau_Hist2D.Scale(signalYield->getValue()/sig_cTau_Hist2D.Integral()*XHScale);
//   bkg_cTau_Hist2D.Scale(HxcTauSB.GetEntries()/bkg_cTau_Hist2D.Integral()*XHScale);
    
  pdf_cTauSTau_X->Scale((signalYield->getValue()+bckgYield->getValue())/pdf_cTauSTau_X->Integral()*XHScale);
  sig_cTauSTau_X->Scale((signalYield->getValue())/sig_cTauSTau_X->Integral()*XHScale);
  bkg_cTauSTau_X->Scale((HxcTauSB.GetEntries())/bkg_cTauSTau_X->Integral()*XHScale);
//  bkg_cTauSTau_X->Scale((bckgYield->getValue())/bkg_cTauSTau_X->Integral()*XHScale);
  pdf_cTauSTau_Y->Scale((signalYield->getValue()+bckgYield->getValue())/pdf_cTauSTau_Y->Integral()*XHScale);
  sig_cTauSTau_Y->Scale((signalYield->getValue())/sig_cTauSTau_Y->Integral()*XHScale);
  bkg_cTauSTau_Y->Scale((HxcTauSB.GetEntries())/bkg_cTauSTau_Y->Integral()*XHScale);
//  bkg_cTauSTau_Y->Scale((bckgYield->getValue())/bkg_cTauSTau_Y->Integral()*XHScale);
  
  

//   sig_STau_Hist.Scale(signalYield->getValue()/sig_STau_Hist.Integral()*XHScale);
//   bkg_STau_Hist.Scale(HxScTauSB.GetEntries()/bkg_STau_Hist.Integral()*XHScale);
//   pdf_STau_Hist.Scale((signalYield->getValue()+bckgYield->getValue())/pdf_STau_Hist.Integral()*XHScale);
//  bkg_cTau_Hist.Scale(bckgYield->getValue()/bkg_cTau_Hist.Integral()*XHScale);
 
//   for (int i = 0; i < xMass.getNumBins(); ++i) {
//     double val = pdfHist.GetBinContent(i+1); 
//     val /= totalPdf; 
//     val *= totalData;
//     pdfHist.SetBinContent(i+1, val); 
//     val = sigHist.GetBinContent(i+1); 
//     val /= totalPdf; 
//     val *= sigFrac->getValue(); 
//     val *= totalData;
//     sigHist.SetBinContent(i+1, val); 
//     val = bkgHist.GetBinContent(i+1); 
//     val /= totalPdf; 
//     val *= (1.0 - sigFrac->getValue());
//     val *= totalData;
//     bkgHist.SetBinContent(i+1, val); 
//   }

//  double sigmaw    = sigma1.getValue()*wg1->getValue()+ (1-wg1->getValue())*sigma2.getValue();
//  double sigmawErr = sqrt(sigma1.getError()*wg1->getValue()*sigma1.getError()*wg1->getValue()+ (1-wg1->getValue())*sigma2.getError()*(1-wg1->getValue())*sigma2.getError());
  c1->cd();
    TLegend* leg_sign = new TLegend(0.30,0.70,0.90,0.90);
    leg_sign->SetTextSize(0.025) ;
    leg_sign->SetTextAlign(31);
    leg_sign->SetBorderSize(0.);
    leg_sign->SetFillStyle(0);
    leg_sign->SetHeader("B^{+} mass spectrum  Fit Projection");
    if(signalYield->getError()!=0){
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Sign} =    %5.0f  #pm %5.0f",signalYield->getValue(),signalYield->getError()),"");
    }else{
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Sign} =    %5.0f Fixed",signalYield->getValue()),"");
    }
    if(bckgYield->getError()!=0){
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Bckg} =    %5.0f  #pm  %5.0f",bckgYield->getValue(),bckgYield->getError()),"");
    }else{
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Bckg} =    %5.0f  Fixed",bckgYield->getValue()),"");
    }
    
    if(mean.getError()!=0){
     leg_sign->AddEntry(&HxMass ,Form( "M_{B^{+}} =   %5.5f  #pm %5.5f",mean.getValue(),mean.getError()),"");
    }else{
     leg_sign->AddEntry(&HxMass ,Form( "M_{B^{+}} =   %5.5f Fixed",mean.getValue()),"");
     }
    if(sigma1.getError()!=0){
     leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{1}_{B^{+}} =   %5.5f  #pm %5.5f",sigma1.getValue(),sigma1.getError()),"");
    }else{
     leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{1}_{B^{+}} =   %5.5f Fixed",sigma1.getValue()),"");
    }
    if(sigma2.getError()!=0){
     leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{2}_{B^{+}} =   %5.5f  #pm %5.5f",sigma2.getValue(),sigma2.getError()),"");
    }else{
     leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{2}_{B^{+}} =   %5.5f Fixed",sigma2.getValue()),"");
    }
  HxMass.GetXaxis()->SetTitle("Mass (GeV/c^{2})");
  HxMass.SetMarkerStyle(8);
  HxMass.SetMarkerSize(MarkerSize);
  HxMass.SetTitle("");
  HxMass.Draw("E1"); 
//  HxMass.Draw("p"); 
  pdfHist.SetLineWidth(PlotLineWidth); 
  pdfHist.SetLineColor(kBlue);
  pdfHist.Draw("same,HIST"); 
  sigHist.SetLineWidth(PlotLineWidth); 
  sigHist.SetLineColor(kMagenta);
  sigHist.SetLineStyle(kDashed); 
  sigHist.Draw("same,HIST"); 
  bkgHist.SetLineWidth(PlotLineWidth); 
  bkgHist.SetLineColor(kRed);
  bkgHist.SetLineStyle(kDashed); 
  bkgHist.Draw("same,HIST"); 
  leg_sign->Draw("same");
  HxMass.Write();
  pdfHist.Write();
  sigHist.Write();
  bkgHist.Write();
//  
  c2->cd();
  c2->SetLogy();
  TLegend* leg_pdfSB = new TLegend(0.50,0.65,0.90,0.90);
  leg_pdfSB->SetTextAlign(12);
  leg_pdfSB->SetHeader("B^{+} proper time Fit Projection");
  leg_pdfSB->SetTextSize(0.025) ;
  leg_pdfSB->SetBorderSize(0.);
  leg_pdfSB->SetFillStyle(0);
  leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[4]{#scale[1.5]{#tau}_{B^{+}}  =  %5.4f #pm %5.4f [ps]}",1/(c_const*cTau.getValue()),cTau.getError()/((c_const*cTau.getValue())*(cTau.getValue())))   ,"");
  if( b1.getError()!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "b1   =  %5.4f #pm %5.4f     ",b1.getValue(),b1.getError())   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "b1   =  %5.4f     Fixed     ",b1.getValue())   ,"");
  }   
//   if( b2->getError()!=0){
//       leg_pdfSB->AddEntry(&HxcTau ,Form( "b2   =  %5.3f #pm %5.3f     ",b2->getValue(),b2->getError())   ,"");
//   }else{      
//       leg_pdfSB->AddEntry(&HxcTau ,Form( "b2   =  %5.3f     Fixed     ",b2->getValue())   ,"");
//   }   
  if( tauSB1.getError()!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB1} =  %5.4f #pm %5.4f     }",1/(tauSB1.getValue()),tauSB1.getError()/((tauSB1.getValue())*(tauSB1.getValue())))   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB1} =  %5.4f	 Fixed	  }",1/(tauSB1.getValue()))   ,"");
  }   
  if( tauSB2.getError()!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.4f #pm %5.4f     }",1/(tauSB2.getValue()),tauSB2.getError()/((tauSB2.getValue())*(tauSB2.getValue())))   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.4f	 Fixed	  }",1/(tauSB2.getValue()))   ,"");
  }   
//   if( tauSB3->getError()!=0){
//       leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.3f #pm %5.3f     }",1/(tauSB3->getValue()),tauSB3->getError()/((tauSB3->getValue())*(tauSB3->getValue())))   ,"");
//   }else{      
//       leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.3f Fixed     }",1/(tauSB3->getValue()))   ,"");
//   }   
  leg_pdfSB->AddEntry(&HxcTau ,"#color[4]{#scale[0.8]{- Fit model	     }}"   ,"");
  leg_pdfSB->AddEntry(&HxcTau ,"#color[6]{#scale[0.8]{- Signal model	     }}"   ,"");
  leg_pdfSB->AddEntry(&HxcTau ,"#color[2]{#scale[0.8]{- Background model on SB}}"   ,"");
  HxcTau.SetMinimum(10);
  HxcTauSB.SetMinimum(10);
  HxcTau.GetXaxis()->SetTitle("ct (cm)");
  HxcTau.SetMarkerStyle(8);
  HxcTau.SetMarkerSize(MarkerSize);
  HxcTau.SetTitle("");
  HxcTau.Draw("E1");
  HxcTauSB.SetMarkerStyle(8);
  HxcTauSB.SetMarkerSize(0.5);
  HxcTauSB.SetMarkerColor(kRed);
  HxcTauSB.Draw("same,E1");
  leg_pdfSB->Draw("same");
/*   pdf_cTau_Hist2D.SetLineColor(kBlue);
  pdf_cTau_Hist2D.SetLineWidth(3); 
  pdf_cTau_Hist2D.Draw("same"); 
  sig_cTau_Hist2D.SetLineColor(kMagenta);
  sig_cTau_Hist2D.SetLineStyle(kDashed); 
  sig_cTau_Hist2D.SetLineWidth(2); 
  sig_cTau_Hist2D.Draw("same"); 
  bkg_cTau_Hist2D.SetLineColor(kRed);
  bkg_cTau_Hist2D.SetLineStyle(kDashed); 
  bkg_cTau_Hist2D.SetLineWidth(2); 
  bkg_cTau_Hist2D.Draw("same"); 
 */  
  pdf_cTauSTau_X->SetLineColor(kBlue);
  pdf_cTauSTau_X->SetLineWidth(PlotLineWidth);
  pdf_cTauSTau_X->Draw("same,HIST");
  sig_cTauSTau_X->SetLineColor(kMagenta);
  sig_cTauSTau_X->SetLineWidth(PlotLineWidth);
  sig_cTauSTau_X->SetLineStyle(kDashed);
  sig_cTauSTau_X->Draw("same,HIST");
  bkg_cTauSTau_X->SetLineColor(kRed);
  bkg_cTauSTau_X->SetLineWidth(PlotLineWidth);
  bkg_cTauSTau_X->SetLineStyle(kDashed);
  bkg_cTauSTau_X->Draw("same,HIST");
//   pdf_cTau_Hist.SetLineColor(kBlue);
//   pdf_cTau_Hist.SetLineWidth(3); 
//   pdf_cTau_Hist.Draw("same"); 
//   sig_cTau_Hist.SetLineColor(kMagenta);
//   sig_cTau_Hist.SetLineStyle(kDashed); 
//   sig_cTau_Hist.SetLineWidth(2); 
//   sig_cTau_Hist.Draw("same"); 
//   bkg_cTau_Hist.SetLineColor(kRed);
//   bkg_cTau_Hist.SetLineStyle(kDashed); 
//   bkg_cTau_Hist.SetLineWidth(2); 
//   bkg_cTau_Hist.Draw("same"); 
  
  
  c3->cd();
  TLegend* leg_pdfUncertainty = new TLegend(0.40,0.47,0.90,0.90);
  leg_pdfUncertainty->SetTextAlign(12);
  leg_pdfUncertainty->SetHeader("B^{+} Uncertainty Fit Projection");
  leg_pdfUncertainty->SetTextSize(0.025) ;
  leg_pdfUncertainty->SetBorderSize(0.);
  leg_pdfUncertainty->SetFillStyle(0);
//  leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "#color[4]{#scale[1.5]{#tau}_{B^{+}}  =  %5.3f #pm %5.3f     }",1/(c_const*cTau->getValue()),cTau->getError()/((c_const*cTau->getValue())*(cTau->getValue())))   ,"");
//   if( b1->getError()!=0){
//       leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "b1   =  %5.3f #pm %5.3f     ",b1->getValue(),b1->getError())   ,"");
//   }else{      
//       leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "b1   =  %5.3f     Fixed     ",b1->getValue())   ,"");
//   }   
//   if( tauUncertainty1->getError()!=0){
//       leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Uncertainty1} =  %5.3f #pm %5.3f     }",1/(c_const*tauUncertainty1->getValue()),tauUncertainty1->getError()/((c_const*tauUncertainty1->getValue())*(tauUncertainty1->getValue())))   ,"");
//   }else{      
//       leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Uncertainty1} =  %5.3f	Fixed	  }",1/(c_const*tauUncertainty1->getValue()))   ,"");
//   }   
//   if( tauUncertainty2->getError()!=0){
//       leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Uncertainty2} =  %5.3f #pm %5.3f     }",1/(c_const*tauUncertainty2->getValue()),tauUncertainty2->getError()/((c_const*tauUncertainty2->getValue())*(tauUncertainty2->getValue())))   ,"");
//   }else{      
//       leg_pdfUncertainty->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Uncertainty2} =  %5.3f	Fixed	  }",1/(c_const*tauUncertainty2->getValue()))   ,"");
//   }   
  leg_pdfUncertainty->AddEntry(&HxScTau ,"#color[4]{#scale[0.8]{- Pdf model	     }}"   ,"");
  leg_pdfUncertainty->AddEntry(&HxScTau ,"#color[6]{#scale[0.8]{- Signal model   }}"   ,"");
  leg_pdfUncertainty->AddEntry(&HxScTau ,"#color[2]{#scale[0.8]{- Background model (on SB events)}}"   ,"");
  if( tauErrSign1.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#tau}_{ErrSign }   =  %5.3e #pm %5.3e     }",1/tauErrSign1.getValue(),tauErrSign1.getError()/(tauErrSign1.getValue()*tauErrSign1.getValue()))   ,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#tau}_{ErrSign }   =  %5.3e   Fixed   }",1/tauErrSign1.getValue())   ,"");
  }   
  if( meanGaussianErrSign1.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.0]{M}_{ErrSign1}  =  %5.3e #pm %5.3e     }",meanGaussianErrSign1.getValue(),meanGaussianErrSign1.getError())   ,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.0]{M}_{ErrSign1}  =  %5.3e   Fixed     }",meanGaussianErrSign1.getValue())   ,"");
  }   
  if( sigmaGaussianErrorSign1.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#sigma}_{ErrSign1}  =  %5.3e #pm %5.3e     }",sigmaGaussianErrorSign1.getValue(),sigmaGaussianErrorSign1.getError())	,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#sigma}_{ErrSign1}  =  %5.3e	Fixed	}",sigmaGaussianErrorSign1.getValue())   ,"");
  }   
  if( meanGaussianErrSign2.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.0]{M}_{ErrSign2}  =  %5.3e #pm %5.3e     }",meanGaussianErrSign2.getValue(),meanGaussianErrSign2.getError())   ,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.0]{M}_{ErrSign2}  =  %5.3e   Fixed     }",meanGaussianErrSign2.getValue())   ,"");
  }   
  if( sigmaGaussianErrorSign2.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#sigma}_{ErrSign2}  =  %5.3e #pm %5.3e     }",sigmaGaussianErrorSign2.getValue(),sigmaGaussianErrorSign2.getError())	,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#sigma}_{ErrSign2}  =  %5.3e	Fixed	}",sigmaGaussianErrorSign2.getValue())   ,"");
  }   
  if( tauErrBckg1.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#tau}_{ErrBckg }   =  %5.3e #pm %5.3e     }",1/tauErrBckg1.getValue(),tauErrBckg1.getError()/(tauErrBckg1.getValue()*tauErrBckg1.getValue()))   ,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#tau}_{ErrBckg }   =  %5.3e   Fixed   }",1/tauErrBckg1.getValue())   ,"");
  }   
  if( meanGaussianErrBckg1.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.0]{M}_{ErrBckg1}  =  %5.3e #pm %5.3e     }",meanGaussianErrBckg1.getValue(),meanGaussianErrBckg1.getError())   ,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.0]{M}_{ErrBckg1}  =  %5.3e   Fixed     }",meanGaussianErrBckg1.getValue())   ,"");
  }   
  if( sigmaGaussianErrorBckg1.getError()!=0){
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#sigma}_{ErrBckg1}  =  %5.3e #pm %5.3e     }",sigmaGaussianErrorBckg1.getValue(),sigmaGaussianErrorBckg1.getError())	,"");
  }else{      
      leg_pdfUncertainty->AddEntry(&HxScTau ,Form( "#color[2]{#scale[1.5]{#sigma}_{ErrBckg1}  =  %5.3e	Fixed	}",sigmaGaussianErrorBckg1.getValue())   ,"");
  }   
  HxScTau.Draw("E1"); 
  HxScTau.GetXaxis()->SetTitle("ct (cm)");
  HxScTau.SetTitle("");
  leg_pdfUncertainty->Draw("same");
  HxScTauSB.SetMarkerStyle(8);
  HxScTauSB.SetMarkerSize(MarkerSize);
  HxScTauSB.SetMarkerColor(kRed);
  HxScTauSB.Draw("same,E1");
  pdf_cTauSTau_Y->SetLineWidth(PlotLineWidth);
  pdf_cTauSTau_Y->SetLineColor(kBlue);
  pdf_cTauSTau_Y->Draw("same,HIST");
  sig_cTauSTau_Y->SetLineWidth(PlotLineWidth);
  sig_cTauSTau_Y->SetLineColor(kMagenta);
  sig_cTauSTau_Y->SetLineStyle(kDashed);
  sig_cTauSTau_Y->Draw("same,HIST");
  bkg_cTauSTau_Y->SetLineWidth(PlotLineWidth);
  bkg_cTauSTau_Y->SetLineColor(kRed);
  bkg_cTauSTau_Y->SetLineStyle(kDashed);
  bkg_cTauSTau_Y->Draw("same,HIST");


//   pdf_STau_Hist.SetLineColor(kBlue);
//   pdf_STau_Hist.SetLineWidth(2); 
//   pdf_STau_Hist.Draw("same"); 
//   sig_STau_Hist.SetLineColor(kMagenta);
//   sig_STau_Hist.SetLineStyle(kDashed); 
//   sig_STau_Hist.SetLineWidth(2); 
//   sig_STau_Hist.Draw("same"); 
//   bkg_STau_Hist.SetLineColor(kRed);
//   bkg_STau_Hist.SetLineStyle(kDashed); 
//   bkg_STau_Hist.SetLineWidth(2); 
//   bkg_STau_Hist.Draw("same"); 
  
  HxcTau.Write();
  HxcTauSB.Write();
  HxScTau.Write();
  HxScTauSB.Write();
  pdf_cTauSTau_Hist2D.Write();
  sig_cTauSTau_Hist2D.Write();
  bkg_cTauSTau_Hist2D.Write();
//   pdf_cTau_Hist2D.Write();
//   sig_cTau_Hist2D.Write();
//   bkg_cTau_Hist2D.Write();
//   pdf_cTau_Hist.Write();
//   sig_cTau_Hist.Write();
//   bkg_cTau_Hist.Write();
//   pdf_STau_Hist.Write();
//   sig_STau_Hist.Write();
//   bkg_STau_Hist.Write();
  pdf_cTauSTau_X->Write();
  pdf_cTauSTau_Y->Write();
  sig_cTauSTau_X->Write();
  sig_cTauSTau_Y->Write();
  bkg_cTauSTau_X->Write();
  bkg_cTauSTau_Y->Write();
  c1->Write();
  c2->Write();
  c3->Write();
  char PDFNameMass[50] = "Bp-Mass-2016.pdf";
  char PDFNamecTau[50] = "Bp-cTau-2016.pdf";
  char PDFNameReso[50] = "Bp-Reso-2016.pdf";
  char testo[130] ;
  sprintf(testo,"mv %s %s.tmp",PDFNameMass,PDFNameMass);
  gSystem->Exec(testo);
  sprintf(testo,"mv %s %s.tmp",PDFNamecTau,PDFNamecTau);
  gSystem->Exec(testo);
  sprintf(testo,"mv %s %s.tmp",PDFNameReso,PDFNameReso);
  gSystem->Exec(testo);
  
  std::cout<<"Tau  [ps]= "<<1/(c_const*cTau.getValue())<<"+/-"<<cTau.getError()/((c_const*cTau.getValue())*(cTau.getValue()))<<std::endl;
  std::cout<<"1/TauSB1 = "<<1/(tauSB1.getValue())<<"+/-"<<tauSB1.getError()/((tauSB1.getValue())*(tauSB1.getValue()))<<std::endl;
  std::cout<<"1/TauSB2 = "<<1/(tauSB2.getValue())<<"+/-"<<tauSB2.getError()/((tauSB2.getValue())*(tauSB2.getValue()))<<std::endl;
//  std::cout<<"1/TauSB3 = "<<1/(tauSB3->getValue())<<"+/-"<<tauSB3->getError()/((tauSB3->getValue())*(tauSB3->getValue()))<<std::endl;

  c1->Print(PDFNameMass);
  c2->Print(PDFNamecTau);
  c3->Print(PDFNameReso);
  OutFile->Close();
  std::cout<<"==========================================" <<std::endl;
  std::cout<<"==========================================" <<std::endl;

  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;
  std::cout<<"==========================================" <<std::endl;
  std::cout<<"==========================================" <<std::endl;
}
